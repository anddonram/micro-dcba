#include "hip/hip_runtime.h"
/*
    ABCD-GPU: Simulating Population Dynamics P systems on the GPU, by DCBA 
    ABCD-GPU is a subproject of PMCGPU (Parallel simulators for Membrane 
                                        Computing on the GPU)   
 
    Copyright (c) 2015  Research Group on Natural Computing, Universidad de Sevilla
    					Dpto. Ciencias de la Computación e Inteligencia Artificial
    					Escuela Técnica Superior de Ingeniería Informática,
    					Avda. Reina Mercedes s/n, 41012 Sevilla (Spain)

    Author: Miguel Ángel Martínez-del-Amor
    
    This file is part of ABCD-GPU.
  
    ABCD-GPU is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    ABCD-GPU is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with ABCD-GPU.  If not, see <http://www.gnu.org/licenses/>. */

/*
 * This GPU simulator for PDP systems was introduced and analysed in the paper:
 * M.A. Martínez-del-Amor, I. Pérez-Hurtado, A. Gastalver-Rubio, A.C. Elster
 * M.J. Pérez-Jiménez. Population Dynamics P systems on CUDA. Proceedings of
 * the 10th Conference on Computational Methods in Systems Biology (CMSB2012),
 * London, 2012. Lecture Notes in Bioinformatics, 7605 (2012), 247-266.
 *
 * And extended for the paper:
 * M.A. Martínez-del-Amor, L.F. Macías-Ramos, L. Valencia-Cabrera, M.J. Pérez-
 * Jiménez. Parallel simulation of Population Dynamics P systems: updates and
 * roadmap. Natural Computing. Accepted.
 */

#include "simulator_gpu_dir.h"

#include "curng_binomial.h"
#include "competition.h"

#include <math.h>
#include <limits.h>
#include <iostream>
#include <timestat.h>
#include <cstdlib>
#include <future>

using namespace std;

#define CU_THREADS 256
#define CU_LOG_THREADS 8

/************************************************************/
/* The following sets how the arrays are indexed on the CPU */
/************************************************************/
#define AD_IDX(o,m) (sim*options->num_environments*esize+env*esize+(m)*msize+(o))
#define NB_IDX (sim*options->num_environments*besize+env*besize+block)
#define NR_P_IDX (sim*(options->num_environments*rpsize+(resize-rpsize))+env*rpsize+r)
#define NR_E_IDX (sim*(options->num_environments*rpsize+(resize-rpsize))+options->num_environments*rpsize+(r-rpsize))
#define CH_IDX(m) (sim*(options->num_environments*options->num_membranes)+env*options->num_membranes+(m))
#define MU_IDX(o,m) (sim*options->num_environments*esize+env*esize+(m)*msize+(o))

/************************************************************/
/* The following sets how the arrays are indexed on the GPU */
/************************************************************/
#define D_AD_IDX(o,m) (sim*options.num_environments*esize+env*esize+(m)*msize+(o))
#define D_NB_IDX(b) (sim*options.num_environments*besize+env*besize+(b))

#define D_NR_P_IDX(r) (sim*(options.num_environments*rpsize+(resize-rpsize))+env*rpsize+(r))
#define D_NR_E_IDX(r) (sim*(options.num_environments*rpsize+(resize-rpsize))+options.num_environments*rpsize+((r)-rpsize))
#define D_CH_IDX(m) (sim*(options.num_environments*options.num_membranes)+env*options.num_membranes+(m))
#define D_MU_IDX(o,m) (sim*options.num_environments*esize+env*esize+(m)*msize+(o))

/**************************************/
/* How to construct collision objects */
/**************************************/
#define EMPTY UINT_MAX
#define IS_EMPTY(o) ((o)==UINT_MAX)
#define OBJECT(obj,membr,mult) ((((mult)&0x7FF)<<20)|(((obj)+(membr)*msize)&0xFFFFF))
#define GET_OBJECT(o) (o&0xFFFFF)
#define OBJECT_COLLISION(init,b,o) ((0x80000000|((init)&0xFFF00000)) | (((b)&0x3FF)<<10) | ((o)&0x3FF))
#define COLLISION_GET_TID(o) (((o)>>10)&0x3FF)
#define COLLISION_GET_OBJ(o) ((o)&0x3FF)
#define IS_COLLISION(o) ((o)>>31)
#define COLLISION(o1,o2) (!(((o1)>>31)^((o2)>>31)) && (((o1)&0xFFFFF) == ((o2)&0xFFFFF)))
#define SET_CONF_MULT(obj,mult) (((obj)&0xFFF00000)|((mult)&0xFFFFF))
#define GET_CONF_MULT(o) (o&0xFFFFF)
#define GET_MULT(o) ((o>>20)&0x7FF)

//Using constant memory to load as symbols results in no real gain (nor loss)
__constant__ _options d_options;
__constant__ _computations d_computations;

/***************************************************************************/

/* Constructor of the class */

Simulator_gpu_dir::Simulator_gpu_dir(PDP_Psystem_REDIX* PDPps,int mode,bool accurate, PDP_Psystem_output* PDPout) {
	this->structures=PDPps->structures;
	this->options=PDPps->options;
	runcomp=(mode>=1);
	this->mode=mode;
	this->accurate=accurate; // use n/d mode for normalization by default
	error=false;
	init();

	// The real out (binary, csv...)
	this->PDPout=PDPout;

	/* Configure the standard output class (debugging purposes only) */
	/* Let keep it here, after init(). The initial configuration is */
	/* backed up there, and d_structures is initialized */
	pdp_out = new PDP_Psystem_redix_out_std_gpuwrapper(PDPps,this->d_structures,runcomp,&counters);
}


/*************/
/* MAIN LOOP */
/*************/

bool Simulator_gpu_dir::run() {

	return step(options->time);
}

bool Simulator_gpu_dir::step(int k){

	pdp_out->print_simulator_process("[2] STARTING THE SIMULATION: Using virtual table (direct) algorithm, with CUDA");

	/* Repeat for each Parallel Simulation Block (PSB) */
	for (uint psb=0; psb < options->num_simulations; psb+=sim_parallel) {
		
		if (sim_parallel > options->num_simulations - psb)
			options->num_parallel_simulations=options->num_simulations - psb;
		
		reset(psb);

		pdp_out->print_sim_range(psb,psb+options->num_parallel_simulations);

		auto handle = std::async(std::launch::async,
						&Simulator_gpu_dir::do_nothing,this);

        /* MAIN LOOP */
		for (uint i=0; i<k; i++) {
            pdp_out->print_step(i);

			if (selection())
				return false;

			if (execution())
				return false;

			//Check consistency and updating errors
			//Each cycle or if its last step
			if ((i+1==k||(i+1)%options->error_cycle==0) && check_step_errors())
				return false;

            pdp_out->print_configuration();
            if ((i+1)%options->cycles==0) {
            	//Wait for possible previous copy to end
            	hipStreamSynchronize(copy_stream);
            	retrieve_copy();

            	//We must copy first
            	hipStreamSynchronize(execution_stream);

            	//Wait for previous write to copy to host
            	handle.wait();

            	retrieve_async(psb);

            	handle = std::async(std::launch::async,
            			&Simulator_gpu_dir::write_async,this,psb,i);

            }

		}

	}

    /* Output profiling information */
	pdp_out->print_profiling_table();


	return true;
}
// The function we want to execute on the new thread.
void Simulator_gpu_dir::write_async(int psb,int i)
{
	unsigned int* output_multiset_pointer=structures->configuration.multiset;
	//Wait until the copy to host has finished
	hipStreamSynchronize(copy_stream);

	//cout << "Writing..." << endl;

	if(options->output_filter!=NULL){

		if(options->GPU_filter){
			output_multiset_pointer=output_multiset;
		}

		//Filtered configuration
		for (uint simu=psb; (simu <psb+sim_parallel) && (simu < options->num_simulations); simu++)
			PDPout->write_configuration_filtered(output_multiset_pointer,structures->configuration.membrane,simu,i+1,structures->stringids.id_objects);

	}else{
		for (uint simu=psb; (simu <psb+sim_parallel) && (simu < options->num_simulations); simu++)
			PDPout->write_configuration(output_multiset_pointer,structures->configuration.membrane,simu,i+1,structures->stringids.id_objects);
	}
	//cout << "Finished writing. Next..." << endl;
}
// Aux function, does nothing
void Simulator_gpu_dir::do_nothing()
{
}

/***************************************************************************/
/***************************************/
/* Auxiliary functions Implementations */
/***************************************/

/* Safely add and mult uint numbers, returns true if overflow */
/*
bool safe_u_add(uint& op1, uint op2){
	uint c_test=op1+op2;
	if (c_test-op2 != op1) 
		return true;
	else op1=c_test;
	
	return false;
}

bool safe_u_mul(uint& op1, uint op2) {
	uint c_test=op1*op2;
	if (c_test/op2 != op1) 
		return true;
	else op1=c_test;	
	
	return false;
}*/



//TODO: Make this member to return a boolean value, to check errors
bool Simulator_gpu_dir::init() {

	checkCudaErrors(hipStreamCreate (&execution_stream));
	checkCudaErrors(hipStreamCreate (&copy_stream));
	/* Set auxiliary sizes info */
	esize=options->num_objects*options->num_membranes;
	msize=options->num_objects;
	bpsize=options->num_rule_blocks;
	besize=options->num_blocks_env+options->num_rule_blocks;
	rpsize=structures->pi_rule_size;
	resize=structures->pi_rule_size+structures->env_rule_size;
	asize=((besize>>ABV_LOG_WORD_SIZE)+1);

	/* Initialize GPU */
	char * def_dev = getenv("DEFAULT_DEVICE");
	unsigned int dev;
	if (def_dev!=NULL)
		hipSetDevice(dev= atoi(def_dev));
	else
		hipSetDevice(dev = gpuGetMaxGflopsDeviceId());
	
	checkCudaErrors(hipGetDeviceProperties(&dev_property, dev));

	/* Calculating the amount of memory, and simulations to handle */
	unsigned int dep_mem;
	
	// GPU only
	unsigned int max_memory_gpu = dev_property.totalGlobalMem;

	// Temporally like this until auto-compression process
	//cutilCondition(options->mem < max_memory_gpu);
	dep_mem=options->num_membranes*options->num_environments*sizeof(CHARGE); //membrane
	dep_mem+=options->num_objects*options->num_membranes*options->num_environments*sizeof(MULTIPLICITY); //multiset
	dep_mem+=(options->num_rule_blocks+options->num_blocks_env)*options->num_environments*sizeof(MULTIPLICITY);//nb
	dep_mem+=((structures->pi_rule_size)*options->num_environments+structures->env_rule_size)*sizeof(MULTIPLICITY);//nr
	dep_mem+=options->num_objects*options->num_membranes*options->num_environments*sizeof(float)*2; //addition
	dep_mem+=asize*options->num_environments*sizeof(ABV_T); // ABV activations
	dep_mem+=(1+options->num_membranes*options->num_environments)*sizeof(uint); //data error
	dep_mem+=curng_sizeof_state(CU_THREADS*options->num_environments); //random data

	dep_mem+=options->num_membranes*options->num_environments*sizeof(CHARGE); // membrane for async copy
	dep_mem+=options->num_objects*options->num_membranes*options->num_environments*sizeof(MULTIPLICITY); //multiset for async copy
	dep_mem+=options->objects_to_output*sizeof(MULTIPLICITY); // filtered multiset

	// Add new data structures depending on the number of simulations

	sim_parallel=gsl_min(options->num_simulations,(((unsigned int) max_memory_gpu*0.8)-options->mem)/dep_mem);
	options->num_parallel_simulations=sim_parallel;


	/* Printing information */

	if (sim_parallel==0) {
		cout << "Error: no enough memory to run even a single simulation!" << endl;
		return false; // TODO: Catch this!
	}

	if (options->verbose>=1)
		cout << "Selected GPU device:" << endl <<
				"=> Device: " << dev << " (" << dev_property.name << "), Multiprocessors=" << dev_property.multiProcessorCount <<
				", Total GPU memory=" << dev_property.totalGlobalMem << endl;
	if (options->verbose>1)
		cout << "Information about required memory and parallel simulations" << endl <<
				"=> Static memory: " << options->mem << endl <<
				"=> Algorithm memory for one simulation: " << dep_mem << endl <<
                "=> Parallel simulations: " << sim_parallel << endl <<
                "=> Total memory: " << options->mem + dep_mem*sim_parallel << endl;

	options->mem+=dep_mem*sim_parallel;
	/************************************/
	/* Initialization of GPU structures */
	/************************************/
	
	/* Create initial configuration */
	ini_cfg = structures->configuration;

	structures->configuration.membrane_size=options->num_membranes*options->num_environments*options->num_simulations;
	checkCudaErrors(hipHostMalloc((void**)&structures->configuration.membrane,structures->configuration.membrane_size*sizeof(CHARGE)));

	structures->configuration.multiset_size = options->num_objects*options->num_membranes*options->num_environments*options->num_simulations;
	checkCudaErrors(hipHostMalloc((void**)&structures->configuration.multiset, structures->configuration.multiset_size*sizeof( MULTIPLICITY)));

	/* Init configurations */
	for (int sim=0; sim<options->num_simulations; sim++) {
		for (int env=0; env<options->num_environments; env++) {
			for (int m=0;m<options->num_membranes; m++) {
				structures->configuration.membrane[CH_IDX(m)]=ini_cfg.membrane[env*options->num_membranes+m];
			}
			for (int o=0;o<options->num_membranes*options->num_objects;o++) {
				structures->configuration.multiset[MU_IDX(o,0)]=ini_cfg.multiset[env*esize+o];
			}
		}
	}	
	
	/* Initialize the new data structure for activation bit vector */
	initialize_abv();
	
	/* Initialize new data structure for data error */
	data_error_size = 1+2*options->num_membranes*options->num_environments*sim_parallel;
	data_error = new uint[data_error_size];
	for (unsigned int i=0; i<data_error_size; i++)
		data_error[i]=0;
	
	/* Initialize aux data on CPU only if necessary */
	structures->nb_size=(options->num_rule_blocks+options->num_blocks_env)*options->num_environments*sim_parallel;
		
	/* Initialize Nb only inf CPU is going to be executed */
	if (runcomp) {
		structures->nb = new MULTIPLICITY [structures->nb_size];	
	} else
		structures->nb = NULL;

	/* Initialize Nr only if the CPU is used, or for verbosity stuff */
	structures->nr_size= ((structures->pi_rule_size)*options->num_environments+structures->env_rule_size)*sim_parallel;
		
	if (options->verbose>1 || runcomp) {	
		structures->nr = new MULTIPLICITY [structures->nr_size]; 
	} else
		structures->nr = NULL;

	/* For printting purposes */
	unsigned int d_nb_size=(options->num_rule_blocks+options->num_blocks_env)*options->num_environments*sim_parallel;
	
	if (options->verbose>1) {
		d_nb = new unsigned int[d_nb_size];
	} else
		d_nb = NULL;
	
	/* Initialize auxiliary structures for normalization */
	addition_size=options->num_objects*options->num_membranes*options->num_environments*sim_parallel;
	bool finished = false;
	
	while (!finished) {
		/* Use n/d notation for row additions */
		if (accurate) {
			/* Check overflows */
			finished=true;
			bool overflow=false;
			
			ini_denominator = new uint[esize];
			ini_numerator = new uint[esize];
			denominator = ini_denominator;
			numerator = new uint [addition_size];
			addition = NULL;
			
			for (int i=0;i<esize;i++) {
				ini_denominator[i]=1;
				ini_numerator[i]=0;
			}
			
			for (uint block=0; block<besize; block++) {
				for (unsigned int o=structures->ruleblock.lhs_idx[block]; o<structures->ruleblock.lhs_idx[block+1]; o++) {
					unsigned int obj=structures->lhs.object[o];
					unsigned int mult=GET_MULTIPLICITY(structures->lhs.mmultiplicity[o]);
					unsigned int membr=GET_MEMBR(structures->lhs.mmultiplicity[o]);
					
					uint a=ini_denominator[membr*options->num_objects+obj];
					uint b=mult;
					
					int multiple1 = a % b;
					int multiple2=1;
					if (multiple1!=0)
						multiple2 = b % a;
					
					/* If a is multiple of b */
					if (multiple1==0) {
						overflow=safe_u_add(ini_numerator[membr*options->num_objects+obj],a/b);
					}
					/* If b is multiple of a */
					else if (multiple2==0) {
						overflow=safe_u_mul(ini_numerator[membr*options->num_objects+obj],b/a);
						overflow=overflow||safe_u_add(ini_numerator[membr*options->num_objects+obj],1);
						ini_denominator[membr*options->num_objects+obj]=b;
					}
					/* If they are no multiple */
					else {
						overflow=safe_u_mul(ini_numerator[membr*options->num_objects+obj],b);
						overflow=overflow||safe_u_add(ini_numerator[membr*options->num_objects+obj],a);
						overflow=overflow||safe_u_mul(ini_denominator[membr*options->num_objects+obj],b);
					}
					if (overflow) break;
				}
				if (overflow) {
					if (options->verbose>0) {
						cout << "Warning: overflow detected in initialization of row sums (accurate mode n/d), switching to float" << endl;
					}
					accurate=false;
					finished=false;
					delete [] ini_denominator;
					delete [] ini_numerator;
					delete [] numerator;
					ini_numerator=numerator=ini_denominator=denominator=NULL;
					break;
				}
			}
		}
		/* Use float notation for row additions */
		else {
			if (runcomp)
				addition = new float[addition_size];
			else
				addition = NULL;
			finished=true;
		}
	}
	
	/* Select a phase2 kernel */
	size_t sh_mem=((CU_THREADS >> ABV_LOG_WORD_SIZE) + 2*CU_THREADS + options->max_lhs*CU_THREADS)*sizeof(uint);
	if (sh_mem > dev_property.sharedMemPerBlock)
		mode=2; // Use generic kernel
			
	
	/************************************/
	/* Initialization of GPU structures */
	/************************************/
	d_structures = new PDP_Psystem_REDIX::struct_structures;
	d_structures->ruleblock_size = structures->ruleblock_size;
	d_structures->env_rule_size = structures->env_rule_size;
	d_structures->lhs_size = structures->lhs_size;
	d_structures->rhs_size = structures->rhs_size;
	d_structures->pi_rule_size = structures->pi_rule_size;
	d_structures->probability_size = structures->probability_size;

	d_structures->configuration.membrane_size = options->num_membranes*options->num_environments*sim_parallel;
	d_structures->configuration.multiset_size = options->num_objects*options->num_membranes*options->num_environments*sim_parallel;
	
	d_structures->nr_size = //(options->num_rule_blocks+options->num_blocks_env)*options->num_environments*sim_parallel;
			(structures->pi_rule_size+structures->env_rule_size)*options->num_environments*sim_parallel;
	d_structures->nb_size = (options->num_rule_blocks+options->num_blocks_env)*options->num_environments*sim_parallel;

	/* Print new information */
	if (options->verbose>1) {
		cout << " => Memory used by the virtual table algorithm on the GPU:" << endl;

		if (accurate) {
			cout << "Denominator: " << esize*sizeof(uint) << " (" << esize*sizeof(uint)/1024 << "KB)" << endl;
			cout << "Numerator: " << addition_size*sizeof(uint) << " (" << addition_size*sizeof(uint)/1024 << "KB)" << endl;
		} else
			cout << "Addition: " << addition_size*sizeof(float) << " (" << addition_size*sizeof(float)/1024 << "KB)" << endl;
		
		cout << "Nb: " << d_structures->nb_size*sizeof(unsigned int) << " (" << d_structures->nb_size*sizeof(unsigned int)/1024 << "KB)" << endl;
		cout << "Nr: " << d_structures->nr_size*sizeof(unsigned int) << " (" << d_structures->nr_size*sizeof(unsigned int)/1024 << "KB)" << endl;
		cout << "ABV: " << abv_size*sizeof(ABV_T) << " (" << abv_size*sizeof(ABV_T)/1024 << "KB)" << endl;
		int rngsize=curng_sizeof_state(CU_THREADS*options->num_environments*sim_parallel);
		cout << "RNG: " << rngsize << " (" << rngsize/1024 << "KB)" << endl;
		cout << "Errors: " << data_error_size*sizeof(uint) << " (" << data_error_size*sizeof(uint)/1024 << "KB)" << endl;
		cout << "Membrane charges: " << structures->configuration.membrane_size*sizeof(char) << " (" << structures->configuration.membrane_size*sizeof(char)/1024 << "KB)" << endl;
		cout << "Multisets: " << structures->configuration.multiset_size*sizeof(unsigned int) << " (" << structures->configuration.multiset_size*sizeof(unsigned int)/1024 << "KB)" << endl;

		int count=0;
		float div=1;
		char unit[6]={' ','K','M','G','T','P'};
		while ((options->mem/div)>1023 && count<3){
			div*=1024;
			count++;
		}
		cout << "TOTAL: " << options->mem << " (" << options->mem/div << " " << unit[count] << "B)" << endl << endl;
	}

	/* Allocation */
	// Allocate Ruleblock
	checkCudaErrors(hipMalloc((void**)&(d_structures->ruleblock.lhs_idx), (d_structures->ruleblock_size+1)*sizeof(LHS_IDX)));
	checkCudaErrors(hipMalloc((void**)&(d_structures->ruleblock.rule_idx), (d_structures->ruleblock_size+1)*sizeof(RULE_IDX)));
	checkCudaErrors(hipMalloc((void**)&(d_structures->ruleblock.membrane), d_structures->ruleblock_size*sizeof(MEMBRANE)));

	// Allocate LHS
	checkCudaErrors(hipMalloc((void**)&(d_structures->lhs.object), d_structures->lhs_size*sizeof(OBJECT)));
	checkCudaErrors(hipMalloc((void**)&(d_structures->lhs.mmultiplicity), d_structures->lhs_size*sizeof(MULTIPLICITY)));
	checkCudaErrors(hipMalloc((void**)&(d_structures->lhs.imultiplicity), d_structures->lhs_size*sizeof(INV_MULTIPLICITY)));

	// Allocate RHS
	checkCudaErrors(hipMalloc((void**)&(d_structures->rhs.object), d_structures->rhs_size*sizeof(OBJECT)));
	checkCudaErrors(hipMalloc((void**)&(d_structures->rhs.mmultiplicity), d_structures->rhs_size*sizeof(MULTIPLICITY)));

	// Allocate Rule
	checkCudaErrors(hipMalloc((void**)&(d_structures->rule.rhs_idx), (d_structures->pi_rule_size+d_structures->env_rule_size+1)*sizeof(RHS_IDX)));

	// Allocate Probability
	checkCudaErrors(hipMalloc((void**)&(d_structures->probability), d_structures->probability_size*sizeof(PROBABILITY)));

	// Allocate Nr
	checkCudaErrors(hipMalloc((void**)&(d_structures->nr), d_structures->nr_size*sizeof(MULTIPLICITY)));

	// Allocate Nb
	checkCudaErrors(hipMalloc((void**)&(d_structures->nb), d_structures->nb_size*sizeof(MULTIPLICITY)));

	// Allocate Configuration
	checkCudaErrors(hipMalloc((void**)&(d_structures->configuration.multiset), d_structures->configuration.multiset_size*sizeof(MULTIPLICITY)));
	checkCudaErrors(hipMalloc((void**)&(d_structures->configuration.membrane), d_structures->configuration.membrane_size*sizeof(CHARGE)));

	//Allocate Aux Configuration for async copy
	checkCudaErrors(hipMalloc((void**)&(d_configuration.multiset), d_structures->configuration.multiset_size*sizeof(MULTIPLICITY)));
	checkCudaErrors(hipMalloc((void**)&(d_configuration.membrane), d_structures->configuration.membrane_size*sizeof(CHARGE)));

	//Allocate filter if any

	if(options->output_filter!=NULL){
		options->GPU_filter=true;
		checkCudaErrors(hipMalloc((void**)&d_output_filter,options->objects_to_output*sizeof(unsigned int)));

		//Allocate compact multisets
		checkCudaErrors(hipMalloc((void**)&d_output_multiset,sim_parallel*options->objects_to_output*sizeof(MULTIPLICITY)));
		checkCudaErrors(hipHostMalloc((void**)&output_multiset,sim_parallel*options->objects_to_output*sizeof(MULTIPLICITY)));
	}


	// Allocate Additions
	if (!accurate)
		checkCudaErrors(hipMalloc((void**)&d_addition,addition_size*sizeof(float)));
	else {
		checkCudaErrors(hipMalloc((void**)&d_denominator,esize*sizeof(uint)));
		checkCudaErrors(hipMalloc((void**)&d_ini_numerator,esize*sizeof(uint)));
		checkCudaErrors(hipMalloc((void**)&d_numerator,addition_size*sizeof(uint)));
	}



	// Allocate ABV
	checkCudaErrors(hipMalloc((void**)&d_abv,abv_size*sizeof(ABV_T)));


	// Allocate Errors
	checkCudaErrors(hipMalloc((void**)&d_data_error,data_error_size*sizeof(uint)));


	// Allocate RNG states
	//Now the kernel is launched in a stream, so it can execute while the rest of structures are copied to memory
	//We must hipStreamSynchronize after all the memory is set
	curng_binomial_init(dim3(options->num_environments,options->num_parallel_simulations),CU_THREADS,execution_stream,options->fast);

	/* Copies */
	//Now they are async with curng_init!!!

	//If miro-DCBA, make partition
	if(options->micro){
		int* partition=new int[options->num_rule_blocks];
		int* trans_partition=new int[options->num_rule_blocks];
		int* alphabet=new int[options->num_objects*options->num_membranes];

		competition::reset_partition(partition,
				alphabet,
				options->num_rule_blocks,
				options->num_objects*options->num_membranes);

		competition::make_partition_2(partition,
					structures->ruleblock.lhs_idx,
					structures->lhs.object,
					alphabet,
					options->num_rule_blocks,
					options->num_objects,
					options->num_membranes,
					structures->lhs.mmultiplicity,
					structures->lhs_size);
		//Counts the number of different competition blocks
		options-> num_partitions=competition::normalize_partition(partition,trans_partition,options->num_rule_blocks);

		if(options->num_partitions==1){
			cout << "Full competition, micro-DCBA may not improve performance..." << endl;
		}

		options->independent_ruleblocks=competition::initialize_partition_structures(trans_partition,
				options->num_partitions,options->num_rule_blocks,
				&accum_offset,&ordered_rules
				);
		competition::reorder_ruleblocks(structures,ordered_rules,options);

		options->num_partitions-=options->independent_ruleblocks;
		//Ruleblocks that competes with other ruleblocks
		int dependent_ruleblocks=options->num_rule_blocks-options->independent_ruleblocks;
		//checkCudaErrors(hipMalloc((void**)&d_partition,dependent_ruleblocks*sizeof(uint)));
		//checkCudaErrors(hipMemcpyAsync(d_partition, ordered_rules, dependent_ruleblocks*sizeof(uint), hipMemcpyHostToDevice,copy_stream));
		for (int i = 0; i < NUM_STREAMS; ++i) { hipStreamCreate(&streams[i]); }



		delete [] partition;
		delete [] trans_partition;
		delete [] alphabet;
	}


	// Set ABV
	checkCudaErrors(hipMemsetAsync(d_abv,0xFF,abv_size*sizeof(ABV_T),copy_stream));

	// Set Errors
	checkCudaErrors(hipMemsetAsync(d_data_error,0,data_error_size*sizeof(uint),copy_stream));

	//Copy filter filter
	if(options->GPU_filter){
		checkCudaErrors(hipMemcpyAsync(d_output_filter, options->output_filter,options->objects_to_output*sizeof(unsigned int), hipMemcpyHostToDevice,copy_stream));
	}
	

	// Copy Ruleblock
	checkCudaErrors(hipMemcpyAsync(d_structures->ruleblock.lhs_idx, structures->ruleblock.lhs_idx, (d_structures->ruleblock_size+1)*sizeof(LHS_IDX), hipMemcpyHostToDevice,copy_stream));
	checkCudaErrors(hipMemcpyAsync(d_structures->ruleblock.rule_idx, structures->ruleblock.rule_idx, (d_structures->ruleblock_size+1)*sizeof(RULE_IDX), hipMemcpyHostToDevice,copy_stream));
	checkCudaErrors(hipMemcpyAsync(d_structures->ruleblock.membrane, structures->ruleblock.membrane, d_structures->ruleblock_size*sizeof(MEMBRANE), hipMemcpyHostToDevice,copy_stream));

	// Copy LHS
	checkCudaErrors(hipMemcpyAsync(d_structures->lhs.object, structures->lhs.object, d_structures->lhs_size*sizeof(OBJECT), hipMemcpyHostToDevice,copy_stream));
	checkCudaErrors(hipMemcpyAsync(d_structures->lhs.mmultiplicity, structures->lhs.mmultiplicity, d_structures->lhs_size*sizeof(MULTIPLICITY), hipMemcpyHostToDevice,copy_stream));
	checkCudaErrors(hipMemcpyAsync(d_structures->lhs.imultiplicity, structures->lhs.imultiplicity, d_structures->lhs_size*sizeof(INV_MULTIPLICITY), hipMemcpyHostToDevice,copy_stream));

	// Copy RHS
	checkCudaErrors(hipMemcpyAsync(d_structures->rhs.object, structures->rhs.object, d_structures->rhs_size*sizeof(OBJECT), hipMemcpyHostToDevice,copy_stream));
	checkCudaErrors(hipMemcpyAsync(d_structures->rhs.mmultiplicity, structures->rhs.mmultiplicity, d_structures->rhs_size*sizeof(MULTIPLICITY), hipMemcpyHostToDevice,copy_stream));

	// Copy Rule
	checkCudaErrors(hipMemcpyAsync(d_structures->rule.rhs_idx, structures->rule.rhs_idx, (d_structures->pi_rule_size+d_structures->env_rule_size+1)*sizeof(RHS_IDX), hipMemcpyHostToDevice,copy_stream));

	// Copy Probability
	checkCudaErrors(hipMemcpyAsync(d_structures->probability, structures->probability, d_structures->probability_size*sizeof(PROBABILITY), hipMemcpyHostToDevice,copy_stream));

	// Copy Additions
	if (accurate) {
		checkCudaErrors(hipMemcpyAsync(d_denominator, ini_denominator, esize*sizeof(uint), hipMemcpyHostToDevice,copy_stream));
		checkCudaErrors(hipMemcpyAsync(d_ini_numerator, ini_numerator, esize*sizeof(uint), hipMemcpyHostToDevice,copy_stream));
	}	



	//Using constant memory to load as symbols results in no real gain (nor loss)
	checkCudaErrors(hipMemcpyToSymbolAsync(HIP_SYMBOL(d_options), options, sizeof(_options),size_t(0),hipMemcpyHostToDevice,copy_stream));

	_computations* computations;
	computations=new _computations;
	computations->besize=options->num_blocks_env+options->num_rule_blocks;
	computations->esize=options->num_objects*options->num_membranes;
	computations->msize=options->num_objects;
	computations->asize=(besize>>ABV_LOG_WORD_SIZE) + 1;
	computations->block_chunks=(besize + CU_THREADS -1)>>CU_LOG_THREADS;
	computations->rpsize=structures->pi_rule_size;
	computations->resize=structures->pi_rule_size+structures->env_rule_size;

	checkCudaErrors(hipMemcpyToSymbolAsync(HIP_SYMBOL(d_computations), computations, sizeof(_computations), size_t(0),hipMemcpyHostToDevice,copy_stream));
	// Create a timer
	sdkCreateTimer(&counters.timer);

	//Final synchronize
	//hipStreamSynchronize(execution_stream);


	return true;
}

void Simulator_gpu_dir::del() {
	if (addition) delete [] addition;
	if (denominator) delete [] denominator;
	if (ini_numerator) delete [] ini_numerator;
	if (numerator) delete [] numerator;
	
	delete [] d_nb;
	PDP_Psystem_REDIX::Configuration aux;
	aux=structures->configuration;
	structures->configuration=ini_cfg;
	checkCudaErrors(hipHostFree(aux.membrane));
	checkCudaErrors(hipHostFree(aux.multiset));

	if (structures->nb) delete []structures->nb;
	if (structures->nr) delete []structures->nr;
	if (abv) delete []abv;
	if (data_error) delete []data_error;
	
	// Deallocate Ruleblocks
	checkCudaErrors(hipFree(d_structures->ruleblock.lhs_idx));
	checkCudaErrors(hipFree(d_structures->ruleblock.rule_idx));
	checkCudaErrors(hipFree(d_structures->ruleblock.membrane));

	// Deallocate LHS
	checkCudaErrors(hipFree(d_structures->lhs.object));
	checkCudaErrors(hipFree(d_structures->lhs.mmultiplicity));
	checkCudaErrors(hipFree(d_structures->lhs.imultiplicity));

	// Deallocate RHS
	checkCudaErrors(hipFree(d_structures->rhs.object));
	checkCudaErrors(hipFree(d_structures->rhs.mmultiplicity));

	// Deallocate Rule
	checkCudaErrors(hipFree(d_structures->rule.rhs_idx));

	// Deallocate Probability
	checkCudaErrors(hipFree(d_structures->probability));

	// Deallocate Nr
	checkCudaErrors(hipFree(d_structures->nr));

	// Deallocate Nb
	checkCudaErrors(hipFree(d_structures->nb));

	// Deallocate Configuration
	checkCudaErrors(hipFree(d_structures->configuration.multiset));
	checkCudaErrors(hipFree(d_structures->configuration.membrane));

	checkCudaErrors(hipFree(d_configuration.multiset));
	checkCudaErrors(hipFree(d_configuration.membrane));


	//Deallocate filter if any
	if(options->output_filter!=NULL){
		checkCudaErrors(hipFree(d_output_filter));
		checkCudaErrors(hipFree(d_output_multiset));
		checkCudaErrors(hipHostFree(output_multiset));
	}

	// Deallocate Additions
	if (!accurate) checkCudaErrors(hipFree(d_addition));
	else {
		checkCudaErrors(hipFree(d_denominator));
		checkCudaErrors(hipFree(d_ini_numerator));
		checkCudaErrors(hipFree(d_numerator));
	}

	// Deallocate ABV
	checkCudaErrors(hipFree(d_abv));
	
	// Deallocate Errors
	checkCudaErrors(hipFree(d_data_error));
	

	//Deallocate partition for micro
	if(options->micro){
		//checkCudaErrors(hipFree(d_partition));
		delete [] accum_offset;
		delete [] ordered_rules;
		cout<<"printmeh"<<endl;

		for (int i = 0; i < NUM_STREAMS; ++i)
		{
			cout<<"print"<<endl;
			hipStreamDestroy(streams[i]);
		}

	}
	checkCudaErrors(hipStreamDestroy(execution_stream));
	checkCudaErrors(hipStreamDestroy(copy_stream));
	// Deallocate RNG states
	curng_binomial_free();	

	sdkDeleteTimer(&counters.timer);
	
	hipDeviceReset();
}

void Simulator_gpu_dir::reset(int sim_ini) {
	checkCudaErrors(hipMemcpyAsync(d_structures->configuration.membrane, structures->configuration.membrane+sim_ini*options->num_environments*options->num_membranes, options->num_parallel_simulations*options->num_environments*options->num_membranes*sizeof(CHARGE), hipMemcpyHostToDevice,copy_stream));
	checkCudaErrors(hipMemcpyAsync(d_structures->configuration.multiset, structures->configuration.multiset+sim_ini*options->num_environments*esize, options->num_parallel_simulations*options->num_environments*esize*sizeof(MULTIPLICITY), hipMemcpyHostToDevice,copy_stream));
	hipStreamSynchronize(copy_stream);
}

__global__ void kernel_output_filter(MULTIPLICITY* d_output_multiset,
									MULTIPLICITY *src_multiset,
									unsigned int *d_output_filter,
									int max_objects,
									int sim_size){
	//Calculate id
	uint tidx=threadIdx.x+blockIdx.x*blockDim.x;

	//Only write if we are not out of bounds
	if(tidx<max_objects){
		//Thread tidx will write to position tidx
		//The object_id to be written at position tidx is stored in d_output_filter
		uint obj_id=d_output_filter[tidx];

		//Get object from proper position taking offset into account
		d_output_multiset[max_objects*blockIdx.y+tidx]=src_multiset[sim_size*blockIdx.y+obj_id];
	}

}

void Simulator_gpu_dir::retrieve_copy() {
	if(options->GPU_filter){
		uint cu_threads=CU_THREADS;
		uint cu_blocksx=options->objects_to_output/cu_threads;
		if(cu_blocksx==0){
			//Less objects than max threads per block
			//use one block and one thread per block
			cu_threads=options->objects_to_output;
			cu_blocksx=1;
		}else if(options->objects_to_output%cu_threads!=0){
			//there are some objects that do not fill into a block
			//Use extra block and keep track of position
			cu_blocksx++;
		}
		uint cu_blocksy=options->num_parallel_simulations;

		kernel_output_filter<<<dim3(cu_blocksx,cu_blocksy),
									cu_threads,0,execution_stream>>>
									(d_output_multiset,
									d_structures->configuration.multiset,
									d_output_filter,
									options->objects_to_output,
									options->num_environments*esize);
	}
	//getLastCudaError("Error copying filtered output device to device");
	else{
		checkCudaErrors(hipMemcpyAsync(d_configuration.membrane, d_structures->configuration.membrane,d_structures->configuration.membrane_size*sizeof(CHARGE), hipMemcpyDeviceToDevice,execution_stream));
		checkCudaErrors(hipMemcpyAsync(d_configuration.multiset, d_structures->configuration.multiset,d_structures->configuration.multiset_size*sizeof(MULTIPLICITY), hipMemcpyDeviceToDevice,execution_stream));
	}
}
void Simulator_gpu_dir::retrieve_async(int sim_ini) {
	if(options->GPU_filter){
		checkCudaErrors(hipMemcpyAsync(output_multiset, d_output_multiset, options->num_parallel_simulations*options->objects_to_output*sizeof(MULTIPLICITY), hipMemcpyDeviceToHost,copy_stream));
	}
	else
	{
		checkCudaErrors(hipMemcpyAsync(structures->configuration.membrane+sim_ini*options->num_environments*options->num_membranes, d_configuration.membrane, options->num_parallel_simulations*options->num_environments*options->num_membranes*sizeof(CHARGE), hipMemcpyDeviceToHost,copy_stream));
		checkCudaErrors(hipMemcpyAsync(structures->configuration.multiset+sim_ini*options->num_environments*esize, d_configuration.multiset, options->num_parallel_simulations*options->num_environments*esize*sizeof(MULTIPLICITY), hipMemcpyDeviceToHost,copy_stream));
	}
}
//Deprecated
void Simulator_gpu_dir::retrieve(int sim_ini) {

	checkCudaErrors(hipMemcpy(structures->configuration.membrane+sim_ini*options->num_environments*options->num_membranes, d_structures->configuration.membrane, options->num_parallel_simulations*options->num_environments*options->num_membranes*sizeof(CHARGE), hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(structures->configuration.multiset+sim_ini*options->num_environments*esize, d_structures->configuration.multiset, options->num_parallel_simulations*options->num_environments*esize*sizeof(MULTIPLICITY), hipMemcpyDeviceToHost));
}

/***************************************************************************/
/***************/
/* MAIN PHASES */
/***************/

int Simulator_gpu_dir::selection(){

	/* PHASE 1: DISTRIBUTION */
	if (!selection_phase1())
		return 1;

	/* PHASE 2: MAXIMALITY */
	if (!selection_phase2())
		return 2;

	/* PHASE 3: PROBABILITY */
	if (!selection_phase3())
		return 3;
	
	return 0;
}


/***************************************************************************/
/*********************/
/* Selection methods */
/*********************/


/*********************************************/
/*********************/
/* Selection Phase 1 */
/*********************/

/*******************************************/
/* Using activation bit vectors on the GPU */
/*******************************************/
__device__ inline bool d_is_active (uint block, uint * abv) {
	return (abv[(block>>ABV_LOG_WORD_SIZE)]
	               >> ((~block)&ABV_DESPL_MASK))
	        & 0x1;
}

__device__ inline void d_deactivate(uint block, uint * abv) {
	atomicAnd(&(abv[(block>>ABV_LOG_WORD_SIZE)]), ~(0x1<<((~block)&ABV_DESPL_MASK)));
}



/*****************************************/
/* Step 1 (filters) of phase1 on the GPU */
/*****************************************/
__global__ void kernel_phase1_filters(
			PDP_Psystem_REDIX::Ruleblock ruleblock,
			PDP_Psystem_REDIX::Configuration configuration,
			PDP_Psystem_REDIX::Lhs lhs,
			PDP_Psystem_REDIX::NR nb,
			struct _options options,
			uint * d_abv,
			uint * d_data_error) {

	uint env=blockIdx.x;
	uint sim=blockIdx.y;
	uint block=threadIdx.x;
	uint besize=options.num_blocks_env+options.num_rule_blocks;
	uint esize=options.num_objects*options.num_membranes;
	uint msize=options.num_objects;
	uint asize=(besize>>ABV_LOG_WORD_SIZE) + 1;
	uint block_chunks=(besize + blockDim.x -1)>>CU_LOG_THREADS;
	extern __shared__ uint sData[];
	uint *s_abv=sData;
	uint *m_c_charges=sData+(blockDim.x>>ABV_LOG_WORD_SIZE);
	uint *m_c_conflicts=m_c_charges+options.num_membranes;
	__shared__ bool c_conflict;//=false;

	bool active=false;

	// TODO: do not assume that number of threads is always > num membranes
	if (threadIdx.x < options.num_membranes) {
		c_conflict=false;
		m_c_charges[threadIdx.x]=4;
		m_c_conflicts[threadIdx.x]=UINT_MAX;
	}

	for (int bchunk=0; bchunk < block_chunks; bchunk++) {
		block=bchunk*blockDim.x+threadIdx.x;

		if (threadIdx.x < (blockDim.x>>ABV_LOG_WORD_SIZE)) {
			s_abv[threadIdx.x]=ABV_INIT_WORD;
		}

		__syncthreads();

		if (block < besize) {
			/*** Filter 1 ***/
			uint membr=ruleblock.membrane[block];

			// Case for rule blocks in Pi
			if (IS_MEMBRANE(membr)) {
				uint am=GET_MEMBRANE(membr);
				char ch=GET_ALPHA(membr);
				// only active those with charge alpha in LHS
				active=(configuration.membrane[sim*options.num_environments*options.num_membranes+env*options.num_membranes+am] == ch);
			}
			// Case for rule blocks for communication, active only if in corresponding environment
			else if (IS_ENVIRONMENT(membr)) {
				active=(GET_ENVIRONMENT(membr)==env);
			}

			/** Filter 2 **/
			if (active) {
				// Using new registers avoid memory accesses on the for loop
				uint o_init=ruleblock.lhs_idx[block];
				uint o_end=ruleblock.lhs_idx[block+1];
				for (int o=o_init; o < o_end; o++) {
					uint obj=lhs.object[o];
					uint membr=lhs.mmultiplicity[o];
					uint mult=GET_MULTIPLICITY(membr);
					membr=GET_MEMBR(membr);

					// Check if we have enough objects to apply the block
					if (configuration.multiset[sim*options.num_environments*esize+env*esize+membr*msize+obj]<mult) {
						active=false;
						//break;
					}
				}
			}

			if (!active)
				d_deactivate(threadIdx.x,s_abv);
			else if (active && IS_MEMBRANE(membr)) {
				uint am=GET_MEMBRANE(membr);
				char chb=GET_BETA(membr);

				char setch= (char) atomicCAS(&m_c_charges[am],4,(uint)chb);
				if (setch!=4 && setch!= chb)
				/*if (m_c_charges[am]==4) {
					m_c_charges[am]=chb;//GET_BETA(membr);
					//printf("S=%d,B=%d,T=%d: (am=%d,beta=%d) -> (charge=%d)\n",sim,block,threadIdx.x,am,chb,m_c_charges[am]);
				}
				else if (m_c_charges[am]!= chb) /*GET_BETA(membr)*/ {
					//printf("!!S=%d,B=%d,T=%d: (am=%d,beta=%d) -> (charge=%d)\n",sim,block,threadIdx.x,am,chb,m_c_charges[am]);
					m_c_conflicts[am] = block;
					c_conflict = true;
				}
			}

			nb[D_NB_IDX(block)] = 0;
		}

		__syncthreads();

		if (threadIdx.x < (blockDim.x>>ABV_LOG_WORD_SIZE)
				&& threadIdx.x < asize-((bchunk*blockDim.x)>>ABV_LOG_WORD_SIZE)) {
			d_abv[sim*options.num_environments*asize+env*asize+((bchunk*blockDim.x)>>ABV_LOG_WORD_SIZE)+threadIdx.x]=s_abv[threadIdx.x];
		}
	}
	
	if (c_conflict && (threadIdx.x < options.num_membranes)) {
		d_data_error[1+sim*options.num_environments*options.num_membranes
		             +env*options.num_membranes+threadIdx.x]=m_c_charges[threadIdx.x];
		d_data_error[1+gridDim.y*options.num_environments*options.num_membranes
		             +sim*options.num_environments*options.num_membranes
		             +env*options.num_membranes+threadIdx.x]=m_c_conflicts[threadIdx.x];

		if (threadIdx.x==0)// && d_data_error[0]!=CONSISTENCY_ERROR)
			d_data_error[0]=CONSISTENCY_ERROR;
	}
	//__syncthreads();
}

/******************************************************************************************************/
/* Atomic Addition operation for floats                                                               *
 * Solution taken from http://forums.nvidia.com/index.php?showtopic=158039&st=0&p=991561&#entry991561 */
/******************************************************************************************************/
__device__ inline void atomicAddf(float* address, float value){
	#if __CUDA_ARCH__ >= 200 // for Fermi, atomicAdd supports floats
		atomicAdd(address,value);
	#elif __CUDA_ARCH__ >= 110
		// float-atomic-add
		float old = value;
		while ((old = atomicExch(address, atomicExch(address, 0.0f)+old))!=0.0f);
	#endif
}


/************************************************************/
/* Step 2 (normalization and minimums) of phase1 on the GPU */
/* This non-accurate version causes updating errors
/************************************************************/
__global__ void kernel_phase1_normalization(
		PDP_Psystem_REDIX::Ruleblock ruleblock,
		PDP_Psystem_REDIX::Configuration configuration,
		PDP_Psystem_REDIX::Lhs lhs,
		PDP_Psystem_REDIX::NR nr,
		struct _options options,
		float * d_addition,
		uint * d_abv,
		uint obj_chunks) {

	uint env=blockIdx.x;
	uint sim=blockIdx.y;
	uint block=threadIdx.x;
	uint besize=options.num_blocks_env+options.num_rule_blocks;
	uint esize=options.num_objects*options.num_membranes;
	uint msize=options.num_objects;
	uint asize=(besize>>ABV_LOG_WORD_SIZE) + 1;
	uint block_chunks=(besize + blockDim.x -1)>>CU_LOG_THREADS;
	extern __shared__ uint s_abv[];

	// Initialize addition vector
	for (int ochunk=0; ochunk < obj_chunks; ochunk++) {
		uint obj=ochunk*blockDim.x+threadIdx.x;
		if (obj>=esize) break;
		d_addition[sim*options.num_environments*esize+env*esize+obj]=1.0f;
	}
	__syncthreads();

	/* Normalization - step 1 *
	 *  calculate the sum of objects in lhs */
	for (int bchunk=0; bchunk < block_chunks; bchunk++) {
		block=bchunk*blockDim.x+threadIdx.x;

		if (block >= besize) break;
		
		if (threadIdx.x < (blockDim.x>>ABV_LOG_WORD_SIZE)
				&& threadIdx.x < asize-((bchunk*blockDim.x)>>ABV_LOG_WORD_SIZE)) {
			s_abv[threadIdx.x]=d_abv[sim*options.num_environments*asize+env*asize+((bchunk*blockDim.x)>>ABV_LOG_WORD_SIZE)+threadIdx.x];
		}
		__syncthreads();
		
		// If the block is activated
//		if((d_abv[sim*options.num_environments*asize+env*asize+(block>>ABV_LOG_WORD_SIZE)]
//			        >> ((~threadIdx.x)&ABV_DESPL_MASK))
//					& 0x1) {
		if (d_is_active(threadIdx.x,s_abv)) {
			uint o_init=ruleblock.lhs_idx[block];
			uint o_end=ruleblock.lhs_idx[block+1];
			for (int o=o_init; o < o_end; o++) {
				uint obj=lhs.object[o];
				uint membr=lhs.mmultiplicity[o];
				//uint mult=GET_MULTIPLICITY(membr);
				membr=GET_MEMBR(membr);

				// TODO: Check if using imultiplicity is more efficient
				float inv=lhs.imultiplicity[o];
				atomicAddf(d_addition+sim*options.num_environments*esize+env*esize+membr*msize+obj,inv);
				// TODO: Replace this for using d / n format
			}
		}
		__syncthreads();
	}
	__syncthreads();

	/* Normalization - step 2 *
	 * Column minimum calculation */
	for (int bchunk=0; bchunk < block_chunks; bchunk++) {
        uint min=0;
		
		block=bchunk*blockDim.x+threadIdx.x;
		if (block >= besize) break;

		if (threadIdx.x < (blockDim.x>>ABV_LOG_WORD_SIZE)
				&& threadIdx.x < asize-((bchunk*blockDim.x)>>ABV_LOG_WORD_SIZE)) {
			s_abv[threadIdx.x]=d_abv[sim*options.num_environments*asize+env*asize+((bchunk*blockDim.x)>>ABV_LOG_WORD_SIZE)+threadIdx.x];
		}
		__syncthreads();
		
		// If the block is activated
//		if((d_abv[sim*options.num_environments*asize+env*asize+(block>>ABV_LOG_WORD_SIZE)]
//			        >> ((~threadIdx.x)&ABV_DESPL_MASK))
//					& 0x1) {
		if (d_is_active(threadIdx.x,s_abv)) {
			min=UINT_MAX;
            uint o_init=ruleblock.lhs_idx[block];
			uint o_end=ruleblock.lhs_idx[block+1];
			for (int o=o_init; o < o_end; o++) {
				uint obj=lhs.object[o];
				uint membr=lhs.mmultiplicity[o];
				uint mult=GET_MULTIPLICITY(membr);
				membr=GET_MEMBR(membr);
				
				//uint value = configuration.multiset[sim*options.num_environments*esize+env*esize+membr*msize+obj]/(mult*mult);
				//value =	value / (d_addition[sim*options.num_environments*esize+env*esize+membr*msize+obj]-1.0f);
				uint value = configuration.multiset[sim*options.num_environments*esize+env*esize+membr*msize+obj]/(mult*mult*(d_addition[sim*options.num_environments*esize+env*esize+membr*msize+obj]-1.0f));
				
				min=(value < min) ? value : min;
				if(min==0) break;
			}
		}
		__syncthreads();
		nr[sim*options.num_environments*besize+env*besize+block]=min;
	}
}

//TODO: Implemented, but not used. I keep it just for interest
__device__ inline void atomicMul(uint* address, uint value){
	#if __CUDA_ARCH__ >= 110
		// atomic-mul
	if (value!=1) {
		uint old = value;
		while ((old = atomicExch(address, atomicExch(address, 1)*old))!=1);
	}
	#endif
}

__global__ void kernel_phase1_normalization_acu (
		PDP_Psystem_REDIX::Ruleblock ruleblock,
		PDP_Psystem_REDIX::Configuration configuration,
		PDP_Psystem_REDIX::Lhs lhs,
		PDP_Psystem_REDIX::NR nr,
		struct _options options,
		uint * d_denominator,
		uint * d_numerator,
		uint * d_ini_numerator,
		uint * d_abv,
		uint obj_chunks) {

	uint env=blockIdx.x;
	uint sim=blockIdx.y;
	uint block=threadIdx.x;
	uint besize=options.num_blocks_env+options.num_rule_blocks;
	uint esize=options.num_objects*options.num_membranes;
	uint msize=options.num_objects;
	uint asize=(besize>>ABV_LOG_WORD_SIZE) + 1;
	uint block_chunks=(besize + blockDim.x -1)>>CU_LOG_THREADS;
	extern __shared__ uint s_abv[];

	// Initialize addition vector
	for (int ochunk=0; ochunk < obj_chunks; ochunk++) {
		uint obj=ochunk*blockDim.x+threadIdx.x;
		if (obj>=esize) break;		
		d_numerator[D_AD_IDX(obj,0)]=d_ini_numerator[obj];
	}
	__syncthreads();

	/* Normalization - step 2 *
	 *  calculate the sum of objects in lhs */
	for (int bchunk=0; bchunk < block_chunks; bchunk++) {
		block=bchunk*blockDim.x+threadIdx.x;
		
		if ((block < besize) && threadIdx.x < (blockDim.x>>ABV_LOG_WORD_SIZE)
				&& threadIdx.x < asize-((bchunk*blockDim.x)>>ABV_LOG_WORD_SIZE)) {
			s_abv[threadIdx.x]=d_abv[sim*options.num_environments*asize+env*asize+((bchunk*blockDim.x)>>ABV_LOG_WORD_SIZE)+threadIdx.x];
		}
		__syncthreads();


		//
		// We start by having the total sum and inactive blocks substract their multiplicities
		if ((block < besize) &&
//				!((d_abv[sim*options.num_environments*asize+env*asize+(block>>ABV_LOG_WORD_SIZE)]
//					        >> ((~threadIdx.x)&ABV_DESPL_MASK))
//							& 0x1)) {
				!d_is_active(threadIdx.x,s_abv)) {
			uint o_init=ruleblock.lhs_idx[block];
			uint o_end=ruleblock.lhs_idx[block+1];
			for (int o=o_init; o < o_end; o++) {
				uint obj=lhs.object[o];
				uint membr=lhs.mmultiplicity[o];
				uint mult=GET_MULTIPLICITY(membr);
				membr=GET_MEMBR(membr);

				atomicSub(d_numerator+D_AD_IDX(obj,membr),d_denominator[membr*options.num_objects+obj]/mult);
			}
		}
		__syncthreads();
	}

	/* Normalization - step 2 *
	 * Column minimum calculation */
	for (int bchunk=0; bchunk < block_chunks; bchunk++) {
        uint min=0;
		
		block=bchunk*blockDim.x+threadIdx.x;
		//if (block >= besize) break;

		if ((block < besize) && threadIdx.x < (blockDim.x>>ABV_LOG_WORD_SIZE)
				&& threadIdx.x < asize-((bchunk*blockDim.x)>>ABV_LOG_WORD_SIZE)) {
			s_abv[threadIdx.x]=d_abv[sim*options.num_environments*asize+env*asize+((bchunk*blockDim.x)>>ABV_LOG_WORD_SIZE)+threadIdx.x];
		}
		__syncthreads();
		
		// If the block is active
		if ((block < besize) &&
//				((d_abv[sim*options.num_environments*asize+env*asize+(block>>ABV_LOG_WORD_SIZE)]
//									        >> ((~threadIdx.x)&ABV_DESPL_MASK))
//											& 0x1)) {
				d_is_active(threadIdx.x,s_abv)) {
			min=UINT_MAX;
            uint o_init=ruleblock.lhs_idx[block];
			uint o_end=ruleblock.lhs_idx[block+1];

			for (int o=o_init; o < o_end; o++) {
				uint obj=lhs.object[o];
				uint membr=lhs.mmultiplicity[o];
				uint mult=GET_MULTIPLICITY(membr);
				membr=GET_MEMBR(membr);
				
				uint value = (configuration.multiset[D_MU_IDX(obj,membr)] * d_denominator[membr*options.num_objects+obj]) / (mult*mult*d_numerator[D_AD_IDX(obj,membr)]);
				min=(value < min) ? value : min;
				if(min==0) break;
			}
		}
		__syncthreads();
		
		if (block < besize)
			nr[D_NB_IDX(block)]=min;
		//sim*options.num_environments*besize+env*besize+block]=min;
	}
	//__syncthreads();
}

/*****************************************************/
/* Step 3 (update and filter 2) of phase1 on the GPU */
/*****************************************************/
__global__ void kernel_phase1_update(
		PDP_Psystem_REDIX::Ruleblock ruleblock,
		PDP_Psystem_REDIX::Configuration configuration,
		PDP_Psystem_REDIX::Lhs lhs,
		PDP_Psystem_REDIX::NR nb,
		PDP_Psystem_REDIX::NR nr,
		struct _options options,
		uint * d_abv,
		uint * d_data_error) {
	
	extern __shared__ uint s_abv[];
	__shared__ bool block_sel;
	bool update_error=false;
	uint block_upd_error=0;
	
	uint env=blockIdx.x;
	uint sim=blockIdx.y;
	uint block=threadIdx.x;
	uint besize=options.num_blocks_env+options.num_rule_blocks;
	uint esize=options.num_objects*options.num_membranes;
	uint msize=options.num_objects;
	uint asize=(besize>>ABV_LOG_WORD_SIZE) + 1;
	uint block_chunks=(besize + blockDim.x -1)>>CU_LOG_THREADS;
	
	/* Deleting LHS *
	 * Adding block applications */
	for (int bchunk=0; bchunk < block_chunks; bchunk++) {
	
		block=bchunk*blockDim.x+threadIdx.x;
		if (block >= besize) break;
		
		uint bapp=nr[D_NB_IDX(block)];
		
		if (bapp>0) {
            if (!block_sel) block_sel=true;
			
			/* Consume LHS */
            uint o_init=ruleblock.lhs_idx[block];
			uint o_end=ruleblock.lhs_idx[block+1];
			for (int o=o_init; o < o_end; o++) {
				uint obj=lhs.object[o];
				uint membr=lhs.mmultiplicity[o];
				uint mult=GET_MULTIPLICITY(membr);
				membr=GET_MEMBR(membr);
                        
				/* Delete block application and check errors */
				if (atomicSub(configuration.multiset+sim*options.num_environments*esize+env*esize+membr*msize+obj,bapp*mult)
					< bapp*mult)
					if (!update_error) update_error=true;
					block_upd_error = 1+block;
					/* Pre-filter: only filter last rules consuming objects */
					//bapp*mult+mult) ;
					//d_deactivate(threadIdx.x,s_abv);
			}

			/* Add applications to block */
			nb[D_NB_IDX(block)]+=bapp;
		}
	}
	
	/** Filter 2 **/
	
	for (int bchunk=0; bchunk < block_chunks; bchunk++) {
	
		block=bchunk*blockDim.x+threadIdx.x;
		if (block >= besize) break;

		if (threadIdx.x < (blockDim.x>>ABV_LOG_WORD_SIZE)
				&& threadIdx.x < asize-((bchunk*blockDim.x)>>ABV_LOG_WORD_SIZE)) {
			s_abv[threadIdx.x]=d_abv[sim*options.num_environments*asize+env*asize+((bchunk*blockDim.x)>>ABV_LOG_WORD_SIZE)+threadIdx.x];
		}
		__syncthreads();

		if (d_is_active(threadIdx.x,s_abv)) {
			// Using new registers avoid memory accesses on the for loop
			uint o_init=ruleblock.lhs_idx[block];
			uint o_end=ruleblock.lhs_idx[block+1];
			for (int o=o_init; o < o_end; o++) {
				uint obj=lhs.object[o];
				uint membr=lhs.mmultiplicity[o];
				uint mult=GET_MULTIPLICITY(membr);
				membr=GET_MEMBR(membr);

				// Check if we have enough objects to apply the block
				if (configuration.multiset[sim*options.num_environments*esize+env*esize+membr*msize+obj]<mult) {
					d_deactivate(threadIdx.x,s_abv);
					break;
				}
			}
		}		
		
		__syncthreads();
		
		if (threadIdx.x < (blockDim.x>>ABV_LOG_WORD_SIZE)
				&& threadIdx.x < asize-((bchunk*blockDim.x)>>ABV_LOG_WORD_SIZE)) {
			d_abv[sim*options.num_environments*asize+env*asize+((bchunk*blockDim.x)>>ABV_LOG_WORD_SIZE)+threadIdx.x]=s_abv[threadIdx.x];
		}
	}
	//Changed: only save error if it was all ok until here (otherwise we would be overwriting, for example, CONSISTENCY_ERROR)
	if (threadIdx.x==0 && update_error && d_data_error[0]==0) {
		d_data_error[1+sim*options.num_environments*options.num_membranes+env*options.num_membranes]=block_upd_error;
		d_data_error[0]=UPDATING_CONFIGURATION_ERROR;
	}
	
}


/************************************************/
/* Implementation of Phase 1 (calls to kernels) */
/************************************************/
bool Simulator_gpu_dir::selection_phase1() {
    
    pdp_out->print_dcba_phase(1);

    pdp_out->print_profiling_dcba_phase("Launching GPU code for phase 1");
	
	/* Create and start timers */
	if (runcomp) {
		counters.timek1gpu = counters.timek2gpu = counters.timek3gpu = 0;
		counters.timek1cpu = counters.timek2cpu = counters.timek3cpu = 10;
	}

	
	/* USING GPU KERNELS */
	uint cu_threads=CU_THREADS;
	uint cu_blocksx=options->num_environments;
	uint cu_blocksy=options->num_parallel_simulations;

	dim3 dimGrid (cu_blocksx, cu_blocksy);
	dim3 dimBlock (cu_threads);
	size_t sh_mem=((cu_threads>>ABV_LOG_WORD_SIZE) + 2*options->num_membranes)*sizeof(uint);
	uint obj_chunks=(esize + cu_threads -1)/cu_threads;

	/* Apply kernel for filters */
	if (runcomp) {
		pdp_out->print_profiling_dcba_microphase_name("Launching kernel for filters");
		sdkResetTimer(&counters.timer);
		sdkStartTimer(&counters.timer);
	}

	kernel_phase1_filters <<<dimGrid,dimBlock,sh_mem,execution_stream>>> (d_structures->ruleblock,
			d_structures->configuration, d_structures->lhs, d_structures->nb, *options,
			d_abv, d_data_error);
	


	if (runcomp) {
		hipStreamSynchronize(execution_stream);
		getLastCudaError("kernel for phase 1 (filters) launch failure");

		sdkStopTimer(&counters.timer);
		counters.timek1gpu=sdkGetTimerValue(&counters.timer);
		pdp_out->print_profiling_dcba_microphase_result(counters.timek1gpu);
	}
	
	
	
	for (int a=0; a<options->accuracy; a++) {
		/* Apply kernel for normalization */
		
		sh_mem=(cu_threads>>ABV_LOG_WORD_SIZE)*sizeof(uint);
		
		if (runcomp) {
			pdp_out->print_profiling_dcba_microphase_name("Launching kernel for normalization");
			sdkResetTimer(&counters.timer);
			sdkStartTimer(&counters.timer);
		}

		if (! accurate)
		kernel_phase1_normalization <<<dimGrid,dimBlock,sh_mem,execution_stream>>> (d_structures->ruleblock,
			d_structures->configuration, d_structures->lhs, d_structures->nr,
			*options,d_addition,d_abv,obj_chunks);
		else
		kernel_phase1_normalization_acu <<<dimGrid,dimBlock,sh_mem,execution_stream>>> (d_structures->ruleblock,
			d_structures->configuration, d_structures->lhs, d_structures->nr,
			*options,d_denominator,d_numerator,d_ini_numerator,d_abv,obj_chunks);
	

			
		if (runcomp) {
			hipStreamSynchronize(execution_stream);
			getLastCudaError("kernel for phase 1 (normalization) launch failure");

			sdkStopTimer(&counters.timer);
			counters.timek2gpu+=sdkGetTimerValue(&counters.timer);
			pdp_out->print_profiling_dcba_microphase_result(counters.timek2gpu);
		}
	
		/* Apply kernel for update and filter 2 */
	
		if (runcomp) {
			pdp_out->print_profiling_dcba_microphase_name("Launching kernel for updating");
			sdkResetTimer(&counters.timer);
			sdkStartTimer(&counters.timer);
		}

		kernel_phase1_update <<<dimGrid,dimBlock,sh_mem,execution_stream>>> (d_structures->ruleblock,
			d_structures->configuration, d_structures->lhs, d_structures->nb,
			d_structures->nr, *options, d_abv, d_data_error);



		if (runcomp) {
			hipStreamSynchronize(execution_stream);
			getLastCudaError("kernel for phase 1 (update) launch failure");

			sdkStopTimer(&counters.timer);
			counters.timek3gpu+=sdkGetTimerValue(&counters.timer);
			pdp_out->print_profiling_dcba_microphase_result(counters.timek3gpu);
		}
		
	}
	
	pdp_out->print_block_selection();

	pdp_out->print_temporal_configuration();


	/**************************************/
	/* PROFILING AND CHECK-OUT PROCEDURES */
	/**************************************/

//	if (options->verbose > 1) {
//		/* RETRIEVING DATA */
//		checkCudaErrors(hipMemcpy(d_nb, d_structures->nb, d_structures->nb_size*sizeof(MULTIPLICITY), hipMemcpyDeviceToHost));
//		print_block_applications(d_nb);
//
//		d_cfg.multiset = new MULTIPLICITY[structures->configuration.multiset_size];
//
//		checkCudaErrors(hipMemcpy(d_cfg.multiset, d_structures->configuration.multiset, d_structures->configuration.multiset_size*sizeof(MULTIPLICITY), hipMemcpyDeviceToHost));
//
//		print_configuration(d_cfg);
//
//		delete [] d_cfg.multiset;
//	}
	
	/*******************************/
	/* REPRODUCING CODE ON THE CPU */
	/*******************************/
	if (runcomp) {
		gold_selection_phase1_acu();
	
		/* Retrieving times */
		counters.timesp1gpu= counters.timek1gpu+counters.timek2gpu+counters.timek3gpu;
	}
	
	if (runcomp && pdp_out->will_print_dcba_phase()) {
		/***************************************/
		/***** TEMPORAL TESTING PROCEDURE ******/
		/***************************************/

		/* Temporal checking addition, only for debugging */
		/*float * debug_addition = new float[addition_size];
		checkCudaErrors(hipMemcpy(debug_addition, d_addition, addition_size*sizeof(float), hipMemcpyDeviceToHost));

		cout << "Checking addition vectors: " ;//<< endl;
		//cout.precision(15);
		double sum=0.0;
		for (unsigned int sim=0; sim < options->num_parallel_simulations; sim++) {
			for (unsigned int env=0; env < options->num_environments; env++) {
				//cout << endl << "Addition vector on GPU (env " << env << "): ";
				for (unsigned int m=0; m < options->num_membranes; m++) {
					for (int o=0;o<options->num_objects;o++) {
						sum+=fabs((debug_addition[AD_IDX(o,m)]-1.0f-addition[AD_IDX(o,m)]));
							//(debug_addition[AD_IDX(o,m)]-1.0f-addition[AD_IDX(o,m)]);
					//if (fabs(addition[AD_IDX(o,m)]-(debug_addition[AD_IDX(o,m)]-1.0f))>0.5) {
					//	cout << "For sim " << sim << ", env " << env << ", membr " << m << ", obj " << o << ":" ;
					//	cout << "CPU=" << addition[AD_IDX(o,m)] << ", GPU=" << debug_addition[AD_IDX(o,m)]-1.0f <<endl;
					//}
					}
				}
			}
		}
		double deviation=sum;//(double)addition_size;

		delete [] debug_addition;
		//cout.precision(15);
		cout << "Deviation: " << deviation << endl;
		*/
		
		/* Temporal checking of numerators and denominators, only debugging*/
		/*
		uint *d_d = new uint[esize];
		uint *d_n = new uint[addition_size];
		
		checkCudaErrors(hipMemcpy(d_nb, d_structures->nb, d_structures->nb_size*sizeof(MULTIPLICITY), hipMemcpyDeviceToHost));
		checkCudaErrors(hipMemcpy(d_d, d_denominator, esize*sizeof(MULTIPLICITY), hipMemcpyDeviceToHost));
		checkCudaErrors(hipMemcpy(d_n, d_numerator, addition_size*sizeof(MULTIPLICITY), hipMemcpyDeviceToHost));

		
		cout.precision(4);
		for (unsigned int sim=0; sim < options->num_simulations; sim++) {
			cout << "For sim " << sim << ":" << endl;
			for (unsigned int env=0; env < options->num_environments; env++) {
				cout << "For env " << env << ":" << endl;
				for (unsigned int block=0; block <besize; block++) {
					//cout << " b_" << block << "{" << GET_MEMBRANE(structures->ruleblock.membrane[block]) << "," << GET_ALPHA(structures->ruleblock.membrane[block]) << "," << GET_BETA(structures->ruleblock.membrane[block]) << "}*" << structures->nb[NB_IDX];
					if (structures->nb[NB_IDX]!=d_nb[NB_IDX]) {
						cout << " b_" << block << "{am=" << GET_MEMBRANE(structures->ruleblock.membrane[block]) << ", a=" << GET_ALPHA(structures->ruleblock.membrane[block]) << ", b=" << GET_BETA(structures->ruleblock.membrane[block]) << "}: ";
						cout << "CPU=" << structures->nb[NB_IDX] << "vs GPU=" << d_nb[NB_IDX] << ", LHS:" << endl;
						for (unsigned int o=structures->ruleblock.lhs_idx[block]; o<structures->ruleblock.lhs_idx[block+1]; o++) {
							unsigned int obj=structures->lhs.object[o];
							unsigned int mult=GET_MULTIPLICITY(structures->lhs.mmultiplicity[o]);
							unsigned int membr=GET_MEMBR(structures->lhs.mmultiplicity[o]);

							uint val=(ini_cfg.multiset[MU_IDX(obj,membr)]*denominator[options->num_objects*membr+obj]) / (mult*mult*numerator[AD_IDX(obj,membr)]);
							cout << "\t[o" << obj << "]_" << membr << "*" << mult << "=(mult=" << ini_cfg.multiset[MU_IDX(obj,membr)] << ", n/d(CPU)=" << numerator[AD_IDX(obj,membr)] << "/" << denominator[membr*options->num_objects+obj] << ", n/d(GPU)=" << d_n[AD_IDX(obj,membr)] << "/" << d_d[options->num_objects*membr+obj] <<", val=" << val << "), ";
							for (unsigned int block2=0; block2 <besize; block2++) {
								if (block==block2) continue;
								for (unsigned int o2=structures->ruleblock.lhs_idx[block2]; o2<structures->ruleblock.lhs_idx[block2+1]; o2++) {
									unsigned int obj2=structures->lhs.object[o2];
									unsigned int mult2=GET_MULTIPLICITY(structures->lhs.mmultiplicity[o2]);
									unsigned int membr2=GET_MEMBR(structures->lhs.mmultiplicity[o2]);

									if (obj==obj2 && membr==membr2)
										cout << "cb_" << block2 << ",o" << obj2 << "*" << mult2 <<", ";
								}
							}
							cout << endl;
						}
						cout << "}" << endl;
					}
				}
				cout << endl;
			}
		}

		delete [] d_d;
		delete [] d_n;*/
	
		/* Checking ABV */
		ABV_T *debug_abv= new ABV_T[abv_size];
		checkCudaErrors(hipMemcpy(debug_abv, d_abv, abv_size*sizeof(ABV_T), hipMemcpyDeviceToHost));

		int count_errors=0;
		for (unsigned int sim=0; sim < options->num_parallel_simulations; sim++) {
			for (unsigned int env=0; env < options->num_environments; env++) {
				for (unsigned int block=0; block<besize; block++) {
					bool is_gpu_active_abv=(debug_abv[sim*options->num_environments*asize+env*asize+(block>>ABV_LOG_WORD_SIZE)]
    			 		               >> ((~block)&ABV_DESPL_MASK))& 0x1;
					bool is_cpu_active_abv = is_active(block,env,sim);

					if (is_gpu_active_abv!=is_cpu_active_abv) {
					//cout << "ABV for GPU-CPU fails: " << block << "-e" << env << "-s" << sim << ": GPU="
					//<< is_gpu_active_abv << " VS CPU=" << is_cpu_active_abv  << endl;
						count_errors++;
					}
				}
			}
		}

		cout << "Checking ABV: there are " << count_errors << " diferences." << endl;

		delete [] debug_abv;
		
		//d_nb=new uint[d_structures->nb_size];
		/* CALCULATING DIFFERENCES */
		checkCudaErrors(hipMemcpy(d_nb, d_structures->nb, d_structures->nb_size*sizeof(MULTIPLICITY), hipMemcpyDeviceToHost));
		int diff=0;
		int s_diff=0;
		int g_diff=0;
		int sig_diff=0;

		for (unsigned int i=0; i < structures->nb_size; i++) {
			if (structures->nb[i]<d_nb[i]) {
				diff++; s_diff++;
			}
			else if (structures->nb[i]>d_nb[i]) {
				diff++; g_diff++;
			}
			int w_diff = structures->nb[i]-d_nb[i];

			if (w_diff < -1 || w_diff > 1)
				sig_diff++;
		}
		delete []d_nb;

		cout << "Checking NB: there are " << diff << " differences, " << s_diff << " smaller, " << g_diff << " bigger, and " << sig_diff << " significative differences." << endl;
	
		/* PRINTING TIMES */
		cout << endl << "Time for kernel_phase1_filters: GPU=" << counters.timek1gpu << " ms, CPU=" << counters.timek1cpu << " ms, speedup=" << counters.timek1cpu/counters.timek1gpu << "x" << endl;
		cout << endl << "Time for kernel_phase1_normalization: GPU=" << counters.timek2gpu << " ms, CPU=" << counters.timek2cpu << " ms, speedup=" << counters.timek2cpu/counters.timek2gpu << "x" << endl;
		cout << endl << "Time for kernel_phase1_update: GPU=" << counters.timek3gpu << " ms, CPU=" << counters.timek3cpu << " ms, speedup=" << counters.timek3cpu/counters.timek3gpu << "x" << endl;
		cout << endl << "Time for phase 1: GPU=" << counters.timesp1gpu << " ms, CPU=" << counters.timesp1cpu << " ms, speedup=" << counters.timesp1cpu/counters.timesp1gpu << "x" << endl;
	}
	
	pdp_out->print_end_profiling_dcba_phase();

	/** END OF PROCEDURE **/
	return true;
}



/*********************************************/
/*********************/
/* Selection Phase 2 */
/*********************/

/*****************************************/
/* Kernel for Phase 2 version 1: generic */
/*****************************************/
__global__ void kernel_phase2_generic(PDP_Psystem_REDIX::Ruleblock ruleblock,
		PDP_Psystem_REDIX::Configuration configuration,
		PDP_Psystem_REDIX::Lhs lhs,
		PDP_Psystem_REDIX::NR nb,
		PDP_Psystem_REDIX::NR nr,
		struct _options options,
		uint * d_abv) {
	
	extern __shared__ uint sData[];
	__shared__ uint next_b,max_b;
	
	uint bdim = blockDim.x - 1;
	uint * s_abv = sData;
	uint * s_blocks = sData+(bdim >> ABV_LOG_WORD_SIZE);
	uint * s_blocks_update = s_blocks + bdim;
	uint env=blockIdx.x;
	uint sim=blockIdx.y;
	uint block=threadIdx.x;
	uint besize=options.num_blocks_env+options.num_rule_blocks;
	uint esize=options.num_objects*options.num_membranes;
	uint msize=options.num_objects;
	uint asize=(besize>>ABV_LOG_WORD_SIZE) + 1;
	uint block_chunks=(besize + bdim -1)>>CU_LOG_THREADS;
	
	/* One extra iteration, the calculation of sblocks and minimums are pipelined */
	for (int bchunk=0; bchunk < block_chunks+1; bchunk++) {

		block=bchunk*bdim+threadIdx.x;

		/* Only first 256 threads will calculate activations */
		if ((threadIdx.x<bdim) && (bchunk<block_chunks)) {
			s_blocks[threadIdx.x]=UINT_MAX;
			if (threadIdx.x < (bdim>>ABV_LOG_WORD_SIZE)
				&& threadIdx.x < asize-((bchunk*bdim)>>ABV_LOG_WORD_SIZE)) {
				s_abv[threadIdx.x]=d_abv[sim*options.num_environments*asize+env*asize+((bchunk*bdim)>>ABV_LOG_WORD_SIZE)+threadIdx.x];
			}
		}		
		/* and thread 257 will do the hard work, iterate and update configuration */
		else if (threadIdx.x==bdim) {
			max_b=next_b;
			next_b=0;
		}
		
		__syncthreads();
		
		/* Simulating a random re-ordering through thread scheduling */
		if ((threadIdx.x<bdim) && (block < besize) && d_is_active(threadIdx.x,s_abv)) {
			s_blocks[atomicInc(&next_b,bdim+2)]=block;
		}
		/* Hard work for thread 257 */
		else if (threadIdx.x==bdim && bchunk>0) {
			for (int b=0; b<max_b; b++) {
				uint min=UINT_MAX;
				
				block=s_blocks_update[b];

				uint o_init=ruleblock.lhs_idx[block];
				uint o_end=ruleblock.lhs_idx[block+1];
				for (int o=o_init; o < o_end; o++) {;
					uint obj=lhs.object[o];
					uint membr=lhs.mmultiplicity[o];
					uint mult=GET_MULTIPLICITY(membr);
					membr=GET_MEMBR(membr);

					uint value=configuration.multiset[sim*options.num_environments*esize+env*esize+membr*msize+obj]/mult;
										
					min=(value < min) ? value : min;
				}
				if (min>0) {
					for (int o=o_init; o < o_end; o++) {
						uint obj=lhs.object[o];
						uint membr=lhs.mmultiplicity[o];
						uint mult=GET_MULTIPLICITY(membr);
						membr=GET_MEMBR(membr);

						configuration.multiset[sim*options.num_environments*esize+env*esize+membr*msize+obj]-=min*mult;
					}
					nb[sim*options.num_environments*besize+env*besize+block]+=min;
				}
			}
		}
		
		__syncthreads();
		
		//TODO: delete this, just for debuggin purposes
		/*if (threadIdx.x < bdim && block < besize && bchunk < block_chunks)
			nr[sim*options.num_environments*besize+env*besize+block] = s_blocks[threadIdx.x];
		*/
		uint* aux=s_blocks;
		s_blocks=s_blocks_update;
		s_blocks_update=aux;
		
		__syncthreads();
		
		
		//__syncthreads();
		
		// TODO: First solution (following KISS methodology): 
		//       ThreadIdx.x==0 will update everything, and rest of threads will
		//       compute next s_blocks
	}
}
__global__ void kernel_phase2_micro_v2(PDP_Psystem_REDIX::Ruleblock ruleblock,
		PDP_Psystem_REDIX::Configuration configuration,
		PDP_Psystem_REDIX::Lhs lhs,
		PDP_Psystem_REDIX::NR nb,
		PDP_Psystem_REDIX::NR nr,
		struct _options options,
		uint * d_abv,
		int part_init,
		int part_end) {

	extern __shared__ uint sData[];
	//Next b counts the number of blocks
	uint part_size=part_end-part_init;
	//BDim is num threads
	uint bdim = blockDim.x;
	//Activation bit vectors: useless because only accessed once
	//volatile uint * s_abv = sData;
	//Rule order
	uint * s_blocks = sData;
	//Active blocks per partition
	__shared__ uint s_next;

	uint env=blockIdx.x;
	uint sim=blockIdx.y;
	uint block=threadIdx.x;

	//Num of ruleblocks and communication rules
	//At most, only num_rule_blocks
	uint besize=options.num_blocks_env+options.num_rule_blocks;
	//Environment size
	uint esize=options.num_objects*options.num_membranes;
	//Membrane size
	uint msize=options.num_objects;
	uint asize=(besize>>ABV_LOG_WORD_SIZE) + 1;

	uint part_chunks=((part_size) + bdim - 1)>>CU_LOG_THREADS;

	if(threadIdx.x==0){
		s_next=0;
	}
	//__syncthreads();

	for (int bchunk=0; bchunk < part_chunks; bchunk++) {
		__syncthreads();
		//TODO:remove this
		int block_idx=bchunk*bdim+threadIdx.x;

		//if(block_idx>=part_size)break;

		block=block_idx+part_init;

		//Get activation bit vectors
//
//		printf("thread %u block %u abv %u\n",threadIdx.x,block,
//				sim*options.num_environments*asize+
//											 env*asize+
//											 ((block%CU_THREADS)>>ABV_LOG_WORD_SIZE));

		//Why shared memory if only used once?
//		s_abv[threadIdx.x]=d_abv[sim*options.num_environments*asize+
//							 env*asize+
//							 ((block%CU_THREADS)>>ABV_LOG_WORD_SIZE)];
//
//		__syncthreads();

//		if (block < options.num_rule_blocks){
//			printf("%u %#x %d\n ",threadIdx.x,s_abv[threadIdx.x],d_is_active(threadIdx.x,s_abv));
//		}

		//Custom activation index
		//Access abv with index threadIdx.x, but use block%CU_THREADS (bdim) as access
		uint bidx=(block%bdim);
		if (block < part_size &&
				(d_abv[sim*options.num_environments*asize+
											 env*asize+
											 (block>>ABV_LOG_WORD_SIZE)]
							               >> ((~bidx)&ABV_DESPL_MASK))
							        & 0x1) {
			s_blocks[atomicInc(&s_next,bdim+2)]=block;
		}
		__syncthreads();

		if(threadIdx.x==0){
		//1. iterate rules in random order previously calculated
		//2. for each rule, calculate minimum applications
		//3. for each rule, update applications and configurations


		uint o_init,o_end;
		int available_rules=s_next;

		for(int i=0;i<available_rules;i++){
			uint apps=UINT_MAX;

			uint next_block=s_blocks[i];

			//Indexes and lhs lengths
			o_init=ruleblock.lhs_idx[next_block];
			o_end=ruleblock.lhs_idx[next_block+1];

			uint obj;
			uint membr;
			uint rule_mult;

			//Get minimum applications
			for (int o=o_init; o < o_end; o++) {
				obj=lhs.object[o];
				membr=lhs.mmultiplicity[o];
				rule_mult = GET_MULTIPLICITY(membr);
				uint conf_mult = configuration.multiset[D_MU_IDX(GET_OBJECT(obj),0)];

				apps=min(apps,conf_mult/rule_mult);

			}
			//Update applications and configurations
			if(apps==0)continue;

			nb[D_NB_IDX(next_block)]+=apps;

			//printf("Rule %u Applications: %u\n",next_block,apps);
			for (int o=o_init; o < o_end; o++) {
				obj=lhs.object[o];
				membr=lhs.mmultiplicity[o];
				rule_mult = GET_MULTIPLICITY(membr);

//				Check if new multiplicity is valid (>0)
//				If substracting an uint results in a bigger number, then it was negative
//				if(configuration.multiset[D_MU_IDX(GET_OBJECT(obj),0)]
//						  <configuration.multiset[D_MU_IDX(GET_OBJECT(obj),0)]-apps*rule_mult)
//									printf("error on phase 2 micro-v2: rule %u \n",next_block);

				configuration.multiset[D_MU_IDX(GET_OBJECT(obj),0)]-=apps*rule_mult;


			}

		}
		s_next=0;
		}

	}


}


/******************************************************/
/* Kernel for Phase 2, version 2: attempt for speedup */
/******************************************************/
__global__ void kernel_phase2_blhs(PDP_Psystem_REDIX::Ruleblock ruleblock,
		PDP_Psystem_REDIX::Configuration configuration,
		PDP_Psystem_REDIX::Lhs lhs,
		PDP_Psystem_REDIX::NR nb,
		PDP_Psystem_REDIX::NR nr,
		struct _options options,
		uint * d_abv) {
	
	extern __shared__ uint sData[];
	__shared__ uint next_b,max_it;
	
	uint bdim = blockDim.x;
	uint * s_abv = sData;
	uint * s_blocks = sData+(bdim >> ABV_LOG_WORD_SIZE);
	uint * s_itorder = s_blocks + bdim;
	uint * s_blhs = s_itorder + bdim;
	
	uint env=blockIdx.x;
	uint sim=blockIdx.y;
	uint block=threadIdx.x;
	uint besize=options.num_blocks_env+options.num_rule_blocks;
	uint esize=options.num_objects*options.num_membranes;
	uint msize=options.num_objects;
	uint asize=(besize>>ABV_LOG_WORD_SIZE) + 1;
	uint block_chunks=(besize + bdim -1)>>CU_LOG_THREADS;
	
	uint o_init,o_end,o_length;
	
	for (int bchunk=0; bchunk < block_chunks; bchunk++) {

		block=bchunk*bdim+threadIdx.x;

		// Initialize s_blocks
		s_blocks[threadIdx.x]=UINT_MAX;
		// Initialize order
		s_itorder[threadIdx.x]=0;
		// Initialize s_blhs
		/*for (int i=0;i<options.max_lhs;i++) {
			s_blhs[threadIdx.x+i*bdim]=EMPTY;
		}*/
		// Initialize s_abv
		if (threadIdx.x < (bdim>>ABV_LOG_WORD_SIZE)
			&& threadIdx.x < asize-((bchunk*bdim)>>ABV_LOG_WORD_SIZE)) {
			s_abv[threadIdx.x]=d_abv[sim*options.num_environments*asize+env*asize+((bchunk*bdim)>>ABV_LOG_WORD_SIZE)+threadIdx.x];
		}
		else if (threadIdx.x==(bdim>>ABV_LOG_WORD_SIZE)) {
			next_b=0;
			max_it=0;
		}
		
		__syncthreads();

		// Simulating a random re-ordering through thread scheduling 
		// TODO: Implement real random order
		if (block < besize && d_is_active(threadIdx.x,s_abv)) {
			s_blocks[atomicInc(&next_b,bdim+2)]=block;

		}
		
		__syncthreads();
		
		// If there are not active blocks in the chunk
		if (next_b==0) continue;
		
		// Initialize s_blhs with objects from active blocks
		if (threadIdx.x<next_b) {

			block=s_blocks[threadIdx.x];

			o_init=ruleblock.lhs_idx[block];
			o_end=ruleblock.lhs_idx[block+1];
			o_length=o_end-o_init;
			for (int o=o_init; o < o_end; o++) {
				uint obj=lhs.object[o];
				uint membr=lhs.mmultiplicity[o];
				uint mult=GET_MULTIPLICITY(membr);
				membr=GET_MEMBR(membr);
				s_blhs[threadIdx.x*options.max_lhs+o-o_init]=
					OBJECT(obj,membr,mult);
			}
			for (int o=o_length; o<options.max_lhs; o++) {
				s_blhs[threadIdx.x*options.max_lhs+o]=EMPTY;
			}
		}
		
		// Initialize order
		//s_itorder[threadIdx.x]=0;
		
		__syncthreads();
		
		// Calculate object collisions
		for (int i=0; i<next_b; i++) {
			if (threadIdx.x > i && threadIdx.x < next_b) {
				for (int o=0; o < o_length; o++) {
					uint tobj=s_blhs[threadIdx.x*options.max_lhs+o];
					for (int o2=0; o2 < options.max_lhs; o2++) {
						uint iobj=s_blhs[i*options.max_lhs+o2];
						if (IS_EMPTY(iobj))
							break;
						if (COLLISION(iobj,tobj)) {
							s_blhs[threadIdx.x*options.max_lhs+o] =
								OBJECT_COLLISION(tobj,i,o2);
							break; // TODO: Check if this is inneficient
						}
					}
				}
			}
			__syncthreads();
		}
		
		// Calculate iteration order
		for (int i=0; i<next_b; i++) {
			if (threadIdx.x==i) {
				for (int o=0; o < o_length; o++) {
					uint obj=s_blhs[threadIdx.x*options.max_lhs+o];
					if (IS_COLLISION(obj)) {
						uint a=s_itorder[threadIdx.x];
						uint b=s_itorder[COLLISION_GET_TID(obj)]+1;
						a = (a>b)? a : b;
						s_itorder[threadIdx.x] = a;
						max_it = (max_it < a)? a : max_it;
					}							
				}
			}
			__syncthreads();
		}
				
		// Upload multiplicities
		if (threadIdx.x<next_b)
		for (int o=0;o<o_length;o++) {
			uint obj=s_blhs[threadIdx.x*options.max_lhs+o];
			if (!IS_COLLISION(obj)) {
				uint mult = configuration.multiset[D_MU_IDX(GET_OBJECT(obj),0)];
				s_blhs[threadIdx.x*options.max_lhs+o]=SET_CONF_MULT(obj,mult);
			}
		}
		
		__syncthreads();
		
		// Calculate minimum applications
		for (int it=0;it<=max_it;it++) {
			if (threadIdx.x<next_b && s_itorder[threadIdx.x]==it) {
				uint min=UINT_MAX;
				uint value=0;
				// Calculate minimums
				for (int o=0; o < o_length; o++) {
					uint obj=s_blhs[threadIdx.x*options.max_lhs+o];
					if (IS_COLLISION(obj)) {
						uint obj2;
						s_blhs[threadIdx.x*options.max_lhs+o] = obj2 =
						SET_CONF_MULT(obj,
							GET_CONF_MULT(s_blhs[COLLISION_GET_TID(obj)*options.max_lhs+COLLISION_GET_OBJ(obj)]));
						s_blhs[COLLISION_GET_TID(obj)*options.max_lhs+COLLISION_GET_OBJ(obj)]=EMPTY;
						//obj=s_blhs[threadIdx.x*options.max_lhs+o];
						obj=obj2;
					}
					value=GET_CONF_MULT(obj)/GET_MULT(obj);
					min = (value < min)? value : min;
				}
				if (min>0) { // TODO: how efficient is without this?
					s_itorder[threadIdx.x]=max_it+min;
					// Update multiplicities
					for (int o=0; o < o_length; o++) {
						uint obj=s_blhs[threadIdx.x*options.max_lhs+o];
						s_blhs[threadIdx.x*options.max_lhs+o] =
							SET_CONF_MULT(obj,
								GET_CONF_MULT(obj)-min*GET_MULT(obj));
					}
				}
			}
			__syncthreads();
		}
		
		// Update nb
		if (threadIdx.x < next_b && s_itorder[threadIdx.x]>max_it) {
			//nb[sim*options.num_environments*besize+env*besize+s_blocks[threadIdx.x]]+=s_itorder[threadIdx.x]-max_it;
			//nb[D_NB_IDX(s_blocks[threadIdx.x])]+=s_itorder[threadIdx.x]-max_it;
			nb[D_NB_IDX(block)]+=s_itorder[threadIdx.x]-max_it;

		}

		__syncthreads();
		
		// Update configuration
		if (threadIdx.x<next_b) {
			for (int o=0; o < o_length; o++) {
				uint obj=s_blhs[threadIdx.x*options.max_lhs+o];
				if (!IS_EMPTY(obj)) {
					configuration.multiset[D_MU_IDX(lhs.object[o+o_init],GET_MEMBR(lhs.mmultiplicity[o+o_init]))]
						=GET_CONF_MULT(obj);

				}
			}
		}
		
		//TODO: delete this, just for debuggin' purposes
		//if (threadIdx.x==0 && block < besize)
			//nr[sim*options.num_environments*besize+env*besize+block] = next_b;//s_blocks[threadIdx.x];
			//nr[sim*options.num_environments*besize+env*besize+threadIdx.x+bchunk] = next_b;//s_blocks[threadIdx.x];
		//if (threadIdx.x==2 && block < besize)
			//nr[sim*options.num_environments*besize+env*besize+threadIdx.x+bchunk] = max_it;//s_blocks[threadIdx.x];
		
		//__syncthreads();
	}
}


/************************************************/
/* Implementation of Phase 2 (calls to kernels) */
/************************************************/
bool Simulator_gpu_dir::selection_phase2(){
	
//	if (options->verbose>0)	cout << endl << "--------------------------" << endl <<	"Launching GPU code for phase 2" << endl;
	pdp_out->print_dcba_phase(2);

    pdp_out->print_profiling_dcba_phase("Launching GPU code for phase 2");
	
	if (runcomp) {
	//counters.timer = 0;
	counters.timek1gpu = counters.timek2gpu = counters.timek3gpu = 0.0f;
	counters.timek1cpu = counters.timek2cpu = counters.timek3cpu = 1.0f; }
	
	/* USING GPU KERNELS */
	uint cu_threads=CU_THREADS;
	uint cu_blocksx=options->num_environments;
	uint cu_blocksy=options->num_parallel_simulations;

	/* Apply kernel for Phase 2 */
	if (runcomp) {
		//	if (options->verbose>0) cout << endl << "Launching kernel for phase 2: ";
		pdp_out->print_profiling_dcba_microphase_name("Launching kernel for phase 2");
		sdkResetTimer(&counters.timer);
		sdkStartTimer(&counters.timer);
	}
	if(options->micro){
		//TODO: sort this and preaccumulate partitions
		if (pdp_out->will_print_dcba_phase())
			cout << "(using micro DCBA kernel)"<<endl;
	
		dim3 dimGrid (cu_blocksx, cu_blocksy);
		dim3 dimBlock (cu_threads);
		size_t sh_mem=(cu_threads)*sizeof(uint);
	
		hipStreamSynchronize(execution_stream);
		getLastCudaError("pre kernel for phase 2 micro launch failure");

		int stream_to_go=0;
		int start_partition=0;
		//Accumulated size
		int partition_size=0;

		//Trick:If a rule has no competition, then it must have been applied as many times as possible,
		//so there is no point in launching a kernel with it
		for(int i=0;i<options->num_partitions;i++){
			int part_size=accum_offset[i+1] - accum_offset[i];

//			cout<<"start_partition "<<start_partition<< endl;
//			cout<<"part_size (accumulated) "<<partition_size <<endl;
//			cout<<"part_size "<<part_size <<endl;

			if(part_size>=cu_threads){
				if(start_partition!=i){
					//there was something already accumulated, launch it
					kernel_phase2_micro_v2 <<<dimGrid,dimBlock,sh_mem,streams[stream_to_go]>>> (d_structures->ruleblock,
							d_structures->configuration, d_structures->lhs, d_structures->nb,
							d_structures->nr, *options, d_abv,
							accum_offset[start_partition],
							accum_offset[i]);

					stream_to_go++;
					if(stream_to_go==NUM_STREAMS)
						stream_to_go=0;

				}

				//Large partition, launch independently
				kernel_phase2_micro_v2 <<<dimGrid,dimBlock,sh_mem,streams[stream_to_go]>>> (d_structures->ruleblock,
						d_structures->configuration, d_structures->lhs, d_structures->nb,
						d_structures->nr, *options, d_abv,
						accum_offset[i],
						accum_offset[i+1]);

				stream_to_go++;
				if(stream_to_go==NUM_STREAMS)
					stream_to_go=0;
				start_partition=i+1;
				partition_size=0;
			}else{
				//Small partition, accumulate parts
				if(part_size+partition_size >=cu_threads||i+1==options->num_partitions){
					//Enough accumulate (or last iteration), launch
					kernel_phase2_micro_v2 <<<dimGrid,dimBlock,sh_mem,streams[stream_to_go]>>> (d_structures->ruleblock,
							d_structures->configuration, d_structures->lhs, d_structures->nb,
							d_structures->nr, *options, d_abv,
							accum_offset[start_partition],
							accum_offset[i+1]);

					stream_to_go++;
					if(stream_to_go==NUM_STREAMS)
						stream_to_go=0;
					start_partition=i+1;
					partition_size=0;
				}else{
					//Accumulate
					partition_size+=part_size;

				}
			}

		}

		for(int i=0;i<NUM_STREAMS;i++){
			hipStreamSynchronize(streams[i]);
		}
		getLastCudaError("kernel for phase 2 micro launch failure");

	}else{
		if (mode==2) {
			if (pdp_out->will_print_dcba_phase())
				cout << "(using basic kernel)"<<endl;

			dim3 dimGrid (cu_blocksx, cu_blocksy);
			dim3 dimBlock (cu_threads+1);
			size_t sh_mem=((cu_threads>>ABV_LOG_WORD_SIZE) + 2*cu_threads)*sizeof(uint);

			kernel_phase2_generic <<<dimGrid,dimBlock,sh_mem,execution_stream>>> (d_structures->ruleblock,
				d_structures->configuration, d_structures->lhs, d_structures->nb,
				d_structures->nr, *options, d_abv);


		}
		else if (mode<2) {
			if (pdp_out->will_print_dcba_phase())
				cout << "(using blhs kernel)"<<endl;

			dim3 dimGrid (cu_blocksx, cu_blocksy);
			dim3 dimBlock (cu_threads);
			size_t sh_mem=((cu_threads>>ABV_LOG_WORD_SIZE) + 2*cu_threads + options->max_lhs*cu_threads)*sizeof(uint);

			kernel_phase2_blhs <<<dimGrid,dimBlock,sh_mem,execution_stream>>> (d_structures->ruleblock,
				d_structures->configuration, d_structures->lhs, d_structures->nb,
				d_structures->nr, *options, d_abv);


		}
	}
	if (runcomp) {
    	hipStreamSynchronize(execution_stream);
    	getLastCudaError("kernel for phase 2 launch failure");

		sdkStopTimer(&counters.timer);
		counters.timesp2gpu=sdkGetTimerValue(&counters.timer);
		pdp_out->print_profiling_dcba_microphase_result(counters.timesp2gpu);
//		if (options->verbose>0) cout << counters.timesp2gpu << "ms." << endl;
	}

	pdp_out->print_block_selection();

	pdp_out->print_temporal_configuration();


	/**************************************/
	/* PROFILING AND CHECK-OUT PROCEDURES */
	/**************************************/

	if (runcomp && pdp_out->will_print_dcba_phase()) {
		/***************************************/
		/***** TEMPORAL TESTING PROCEDURE ******/
		/***************************************/
		
		/* Checking ABV */
		pdp_out->print_profiling_dcba_microphase_name("Checking maximality");
		ABV_T *debug_abv= new ABV_T[abv_size];
		
		checkCudaErrors(hipMemcpy(debug_abv, d_abv, abv_size*sizeof(ABV_T), hipMemcpyDeviceToHost));

		d_cfg.multiset = new MULTIPLICITY[structures->configuration.multiset_size];
				
		checkCudaErrors(hipMemcpy(d_cfg.multiset, d_structures->configuration.multiset, d_structures->configuration.multiset_size*sizeof(MULTIPLICITY), hipMemcpyDeviceToHost));

		//checkCudaErrors(hipMemcpy(structures->nr, d_structures->nr, d_structures->nr_size*sizeof(MULTIPLICITY), hipMemcpyDeviceToHost));

		checkCudaErrors(hipMemcpy(d_nb, d_structures->nb, d_structures->nb_size*sizeof(MULTIPLICITY), hipMemcpyDeviceToHost));
		
//		print_block_applications(d_nb);
//
//		print_configuration(d_cfg);
		
		
		/*for (unsigned int sim=0; sim < options->num_parallel_simulations; sim++) {
			cout << "Sim " << sim;
			for (unsigned int env=0; env<options->num_environments; env++) {
				cout << endl;
				cout << ", Env " << env;
				cout << ". Next_b: " << structures->nr[sim*options->num_environments*besize+env*besize] <<
					", " << structures->nr[sim*options->num_environments*besize+env*besize+1];
				cout << ". Max_it: " << structures->nr[sim*options->num_environments*besize+env*besize+2] <<
					", " << structures->nr[sim*options->num_environments*besize+env*besize+3];
			}
			cout << endl;
		}*/
		
		/* Check maximality from the GPU */
		uint num_ap_b=0;
		uint num_abv_b=0;
		for (unsigned int sim=0; sim < options->num_parallel_simulations; sim++) {
			for (unsigned int env=0; env < options->num_environments; env++) {
				for (unsigned int block=0; block<besize; block++) {
					bool is_block_active=(debug_abv[sim*options->num_environments*asize+env*asize+(block>>ABV_LOG_WORD_SIZE)]
						       >> ((~block)&ABV_DESPL_MASK))& 0x1;
					if (!is_block_active) continue;

					num_abv_b++;
					bool applicable=true;
					uint o_init=structures->ruleblock.lhs_idx[block];
					uint o_end=structures->ruleblock.lhs_idx[block+1];
					for (uint o=o_init; o < o_end; o++) {
						uint obj=structures->lhs.object[o];
						uint membr=structures->lhs.mmultiplicity[o];
						uint mult=GET_MULTIPLICITY(membr);
						membr=GET_MEMBR(membr);

						// Check if we have enough objects to apply the block
						if (d_cfg.multiset[MU_IDX(obj,membr)]<mult)
							applicable=false;
					}
					if (applicable) {
						num_ap_b++;
						cout << "Error: at sim " << sim << ", env " << env << ", block:" << block;
						uint min=UINT_MAX,val=0;
						for (uint o=o_init; o < o_end; o++) {
							uint obj=structures->lhs.object[o];
							uint membr=structures->lhs.mmultiplicity[o];
							uint mult=GET_MULTIPLICITY(membr);
							membr=GET_MEMBR(membr);

							// Check if we have enough objects to apply the block
							//if (d_cfg.multiset[MU_IDX(obj,membr)]>=mult)
							cout << " [obj" << obj << "]" << membr << "*" << mult << "--" << d_cfg.multiset[MU_IDX(obj,membr)];
							
							val=d_cfg.multiset[MU_IDX(obj,membr)]/mult;
							min=(val<min)?val:min;
						}
						cout << " ==> " << min << " apps" << endl;
					}
				}
			}
		}
		delete []debug_abv;
		delete []d_cfg.multiset;

		pdp_out->print_profiling_dcba_microphase_result(num_ap_b==0 && num_abv_b==0);

		if (num_ap_b>0 || num_abv_b>0)
			cout << "Error from phase2 at GPU: we have " << num_ap_b << " block applications still to perform ("<< num_abv_b << " from abv active blocks)"  << endl;
	}
	
	
	/*******************************/
	/* REPRODUCING CODE ON THE CPU */
	/*******************************/
	if (runcomp)
		gold_selection_phase2();
	
	if (runcomp && pdp_out->will_print_dcba_phase()) {
		// Checking maximality from CPU
		uint num_ap_b=0;
		uint num_abv_b=0;
		for (unsigned int sim=0; sim < options->num_parallel_simulations; sim++) {
			for (unsigned int env=0; env < options->num_environments; env++) {
				for (unsigned int block=0; block<besize; block++) {
					if (! is_active(block,env,sim)) continue;
					num_abv_b++;
					bool applicable=true;
					uint o_init=structures->ruleblock.lhs_idx[block];
					uint o_end=structures->ruleblock.lhs_idx[block+1];
					for (uint o=o_init; o < o_end; o++) {
						uint obj=structures->lhs.object[o];
						uint membr=structures->lhs.mmultiplicity[o];
						uint mult=GET_MULTIPLICITY(membr);
						membr=GET_MEMBR(membr);

						// Check if we have enough objects to apply the block
						if (structures->configuration.multiset[MU_IDX(obj,membr)]<mult)
							applicable=false;
					}
					if (applicable)
						num_ap_b++;
				}
			}
		}

		if (num_ap_b>0 || num_abv_b>0)
			cout << "Error from phase2 at CPU: we have " << num_ap_b << " block applications still to perform (" << num_abv_b << " from abv active blocks)"  << endl;

		// Printing times
		cout << endl << "Time for phase 2: GPU=" << counters.timesp2gpu << " ms, CPU=" << counters.timesp2cpu << " ms, speedup=" << counters.timesp2cpu/counters.timesp2gpu << "x" << endl;
	}
	
	pdp_out->print_end_profiling_dcba_phase();

	return true;
}


/*********************************************/
/*********************/
/* Selection Phase 3 */
/*********************/

/**********************/
/* Kernel for Phase 3 */
/**********************/
__global__ void kernel_phase3(PDP_Psystem_REDIX::Ruleblock ruleblock,
		PDP_Psystem_REDIX::Configuration configuration,
		PDP_Psystem_REDIX::NR nb,
		PDP_Psystem_REDIX::NR nr,
		PDP_Psystem_REDIX::Probability probability//,
//		uint rpsize,
//		uint resize,
//		struct _options options
		) {
	volatile uint env=blockIdx.x;
	volatile uint rpsize=d_computations.rpsize;
	volatile uint resize=d_computations.resize;
	volatile _options options=d_options;
	volatile uint sim=blockIdx.y;
	volatile uint block=threadIdx.x;
	volatile uint besize=d_computations.besize;//options.num_blocks_env+options.num_rule_blocks;
	volatile uint block_chunks=d_computations.block_chunks;//(besize + blockDim.x -1)>>CU_LOG_THREADS;

	for (int bchunk=0; bchunk < block_chunks; bchunk++) {

		block=bchunk*blockDim.x+threadIdx.x;
		
		if (block >= besize) break;
		
		int rule_ini=ruleblock.rule_idx[block];
		int rule_end=ruleblock.rule_idx[block+1];
		
		uint N=0;//nb[D_NB_IDX(block)];
		uint membr=ruleblock.membrane[block];

		if (block<options.num_rule_blocks || env==GET_ENVIRONMENT(membr))
			N=nb[D_NB_IDX(block)];
		
		if (N==0) {
			for (uint r = rule_ini; r < rule_end; r++) {
				if (block < options.num_rule_blocks)
					nr[D_NR_P_IDX(r)] = 0;
				else if (env==GET_ENVIRONMENT(membr))
					nr[D_NR_E_IDX(r)] = 0;
			}
		}
		else {
			//Alternative version along with the memset
		//if(N>0){
			// Update charges
			configuration.membrane[D_CH_IDX(GET_MEMBRANE(membr))]=GET_BETA(membr);
			
			float cr=0.0f,d=1.0f;
			uint r;
			float p;
			uint val=0;
			//Only n-1 rules, to avoid branching on last
			for (r = rule_ini; r < rule_end-1; r++) {
				val=0;
				if (IS_ENVIRONMENT(membr)) {
					p=probability[options.num_environments*rpsize+(r-rpsize)];
				}
				else {
					p=probability[env*rpsize+r];
				}

				cr = fdividef(p,d);
				
				if (cr > 0.0f) {
					val=curng_binomial_random (N, cr);
				}

				if (!IS_ENVIRONMENT(membr))
					nr[D_NR_P_IDX(r)] = val;
				else
					nr[D_NR_E_IDX(r)] = val;

				N-=val;
				d*=(1-cr);
			}

			//Last rule, to avoid one branch on the loop
			r=rule_end-1;
			val=0;
			if (IS_ENVIRONMENT(membr)) {
				p=probability[options.num_environments*rpsize+(r-rpsize)];
			}
			else {
				p=probability[env*rpsize+r];
			}

			cr = fdividef(p,d);

			if (cr > 0.0f) {
				val=N;
			}
			if (!IS_ENVIRONMENT(membr))
				nr[D_NR_P_IDX(r)] = val;
			else
				nr[D_NR_E_IDX(r)] = val;


		}
		__syncthreads();
	}
}

/************************************************/
/* Implementation of Phase 3 (calls to kernels) */
/************************************************/
bool Simulator_gpu_dir::selection_phase3() {

//	if (options->verbose>0) { cout << endl << "--------------------------" << endl; cout << "Launching GPU code for phase 3" << endl; }
	pdp_out->print_dcba_phase(3);

	pdp_out->print_profiling_dcba_phase("Launching GPU code for phase 3");
	
	/* USING GPU KERNELS */
	uint cu_threads=CU_THREADS;
	uint cu_blocksx=options->num_environments;
	uint cu_blocksy=options->num_parallel_simulations;

	/* Apply kernel for Phase 3 */
	
	if (runcomp) {
		pdp_out->print_profiling_dcba_microphase_name("Launching kernel for phase 3");
//		if (options->verbose>0)	cout << endl << "Launching kernel for phase 3: ";
		sdkResetTimer(&counters.timer);
		sdkStartTimer(&counters.timer);
	}

	dim3 dimGrid (cu_blocksx, cu_blocksy);
	dim3 dimBlock (cu_threads);

	//hipMemsetAsync(d_structures->nr,0,d_structures->nr_size*sizeof(MULTIPLICITY),execution_stream);
	kernel_phase3 <<<dimGrid,dimBlock,0,execution_stream>>> (d_structures->ruleblock,
		d_structures->configuration, d_structures->nb, 
		d_structures->nr, d_structures->probability//, d_structures->pi_rule_size,
		//d_structures->pi_rule_size+d_structures->env_rule_size,*options
		);
	 

	if (runcomp) {
		hipStreamSynchronize(execution_stream);
		getLastCudaError("kernel for phase 3 launch failure");

		sdkStopTimer(&counters.timer);
		counters.timesp3gpu=sdkGetTimerValue(&counters.timer);
//		if (options->verbose>0)	cout << counters.timesp3gpu << "ms." << endl;
		pdp_out->print_profiling_dcba_microphase_result(counters.timesp3gpu);
	}
	
	pdp_out->print_rule_selection();


	/**************************************/
	/* PROFILING AND CHECK-OUT PROCEDURES */
	/**************************************/

	/* if (options->verbose>1) {
		// Temporal testing procedure
	
		checkCudaErrors(hipMemcpy(structures->nr, d_structures->nr, d_structures->nr_size*sizeof(MULTIPLICITY), hipMemcpyDeviceToHost));

		checkCudaErrors(hipMemcpy(d_nb, d_structures->nb, d_structures->nb_size*sizeof(MULTIPLICITY), hipMemcpyDeviceToHost));
	
		print_block_applications(d_nb);
		cout << "Checking data from GPU:" << endl;
		for (unsigned int sim=0; sim < options->num_parallel_simulations; sim++)
			for (unsigned int env=0; env<options->num_environments; env++)
				for (unsigned int block=0; block<besize; block++) {
					int rule_ini=structures->ruleblock.rule_idx[block];
					int rule_end=structures->ruleblock.rule_idx[block+1];

					unsigned int N=d_nb[NB_IDX];

					cout << "Sim=" << sim << ", env=" << env << ", block=" << block << ", N=" << N << ":";

					for (unsigned int r = rule_ini; r < rule_end; r++) {
						float p=0.0;

						if (r<rpsize) {
							p=structures->probability[options->num_environments*rpsize+(r-rpsize)];
							cout << "r_" << r-rule_ini << "(p=" << p << ",n=" << structures->nr[NR_P_IDX] << "), ";
						}
						else if (GET_ENVIRONMENT(structures->ruleblock.membrane[block])==env) {
							p=structures->probability[env*rpsize+r];
							cout << "r_" << r-rule_ini << "(p=" << p << ",n=" << structures->nr[NR_E_IDX] << "), ";
						}
					}
					cout << endl;
				}
	}*/
		
	
	/*******************************/
	/* REPRODUCING CODE ON THE CPU */
	/*******************************/
	if (runcomp)
		gold_selection_phase3();
	
	if (runcomp && pdp_out->will_print_rule_selection()) {
		cout << "Checking data from CPU:" << endl;
		for (unsigned int sim=0; sim < options->num_parallel_simulations; sim++)
			for (unsigned int env=0; env<options->num_environments; env++)
				for (unsigned int block=0; block<besize; block++) {
					int rule_ini=structures->ruleblock.rule_idx[block];
					int rule_end=structures->ruleblock.rule_idx[block+1];

					unsigned int N=structures->nb[NB_IDX];

					cout << "Sim=" << sim << ", env=" << env << ", block " << block << ", N=" << N << ":";

					for (unsigned int r = rule_ini; r < rule_end; r++) {
						float p=0.0;

						if (r<rpsize) {
							p=structures->probability[options->num_environments*rpsize+(r-rpsize)];
							cout << "r_" << r-rule_ini << "(p=" << p << ",n=" << structures->nr[NR_P_IDX] << "), ";
						}
						else if (GET_ENVIRONMENT(structures->ruleblock.membrane[block])==env) {
							p=structures->probability[env*rpsize+r];
							cout << "r_" << r-rule_ini << "(p=" << p << ",n=" << structures->nr[NR_E_IDX] << "), ";
						}
					}
					cout << endl;
				}
	}
	
	if (runcomp && pdp_out->will_print_dcba_phase())
		cout << endl << "Time for phase 3: GPU=" << counters.timesp3gpu << " ms, CPU=" << counters.timesp3cpu << " ms, speedup=" << counters.timesp3cpu/counters.timesp3gpu << "x" << endl;
	
	pdp_out->print_end_profiling_dcba_phase();

	return true;
}

/*********************************************/
/***********************/
/* Execution (Phase 4) */
/***********************/

/**********************/
/* Kernel for Phase 4 */
/**********************/
__global__ void kernel_phase4 (PDP_Psystem_REDIX::Rule rule,
			PDP_Psystem_REDIX::Configuration configuration,
			PDP_Psystem_REDIX::Rhs rhs,
			PDP_Psystem_REDIX::NR nr,
			uint rpsize,
			uint resize,
			uint re_chunk,
			struct _options options) {
	
	uint env=blockIdx.x;
	uint sim=blockIdx.y;
	uint r=threadIdx.x;
	uint esize=options.num_objects*options.num_membranes;
	uint msize=options.num_objects;
	uint rp_chunks=(rpsize + blockDim.x -1)>>CU_LOG_THREADS;

	/* Rules of Pi, executed by each environment */
	for (int rchunk=0; rchunk < rp_chunks; rchunk++) {
		r=rchunk*blockDim.x+threadIdx.x;
		
		uint N=0;
		
		if (r < rpsize)
			N=nr[D_NR_P_IDX(r)];
		
		if (N>0) {	
			int o_ini=rule.rhs_idx[r];
			int o_end=rule.rhs_idx[r+1];
			
			for (int o=o_ini; o<o_end; o++) {
				uint obj=rhs.object[o];
				uint mult=rhs.mmultiplicity[o];
				uint membr=GET_MEMBR(mult);
				mult=GET_MULTIPLICITY(mult);

				atomicAdd(&(configuration.multiset[D_MU_IDX(obj,membr)]),N*mult);
			}			
		}
		//__syncthreads();
	}

	/* Communication rules, distributed among the environments */
	uint reini=rpsize+env*re_chunk;
	uint reend=rpsize+(env+1)*re_chunk;
	uint it=0;
	
	r = reini+(it++)*blockDim.x+threadIdx.x;
	
	while ((r<resize) && (r<reend)) {
		int o_ini=rule.rhs_idx[r];
		int o_end=rule.rhs_idx[r+1];

		uint N=nr[D_NR_E_IDX(r)];

		if (N>0)
		for (int o=o_ini; o<o_end; o++) {
			uint obj=rhs.object[o];
			uint denv=rhs.mmultiplicity[o];
			
			obj=sim*options.num_environments*esize+
				denv*esize+obj;
			
			atomicAdd(&(configuration.multiset[obj]),N);
		}
		r = reini+(it++)*blockDim.x+threadIdx.x;
	}
}
__global__ void kernel_phase4_rules (PDP_Psystem_REDIX::Rule rule,
			PDP_Psystem_REDIX::Configuration configuration,
			PDP_Psystem_REDIX::Rhs rhs,
			PDP_Psystem_REDIX::NR nr,
		//	struct _options options,
			int part_init,
			int part_end) {
	_options options=d_options;
	uint resize=d_computations.resize;
	uint rpsize=part_end-part_init;
	uint env=blockIdx.x;
	uint sim=blockIdx.y;
	uint r=threadIdx.x;
	uint esize=options.num_objects*options.num_membranes;
	uint msize=options.num_objects;
	uint rp_chunks=(rpsize + blockDim.x -1)>>CU_LOG_THREADS;

	/* Rules of Pi, executed by each environment */
	for (int rchunk=0; rchunk < rp_chunks; rchunk++) {
		r=rchunk*blockDim.x+threadIdx.x+part_init;

		uint N=0;

		if (r < rpsize)
			N=nr[D_NR_P_IDX(r)];

		if (N>0) {
			int o_ini=rule.rhs_idx[r];
			int o_end=rule.rhs_idx[r+1];

			for (int o=o_ini; o<o_end; o++) {
				uint obj=rhs.object[o];
				uint mult=rhs.mmultiplicity[o];
				uint membr=GET_MEMBR(mult);
				mult=GET_MULTIPLICITY(mult);

				atomicAdd(&(configuration.multiset[D_MU_IDX(obj,membr)]),N*mult);
			}
		}
		//__syncthreads();
	}

}
__global__ void kernel_phase4_env (PDP_Psystem_REDIX::Rule rule,
			PDP_Psystem_REDIX::Configuration configuration,
			PDP_Psystem_REDIX::Rhs rhs,
			PDP_Psystem_REDIX::NR nr,
			uint re_chunk) {
	_options options=d_options;
	uint resize=d_computations.resize;
	uint rpsize=d_computations.rpsize;

	uint env=blockIdx.x;
	uint sim=blockIdx.y;
	uint r=threadIdx.x;
	uint esize=options.num_objects*options.num_membranes;
	uint reini=rpsize+env*re_chunk;
	uint reend=rpsize+(env+1)*re_chunk;
	uint it=0;

	r = reini+(it++)*blockDim.x+threadIdx.x;


	while ((r<resize) && (r<reend)) {
		int o_ini=rule.rhs_idx[r];
		int o_end=rule.rhs_idx[r+1];

		uint N=nr[D_NR_E_IDX(r)];

		if (N>0)
		for (int o=o_ini; o<o_end; o++) {
			uint obj=rhs.object[o];
			uint denv=rhs.mmultiplicity[o];

			obj=sim*options.num_environments*esize+
				denv*esize+obj;

			atomicAdd(&(configuration.multiset[obj]),N);
		}
		r = reini+(it++)*blockDim.x+threadIdx.x;
	}
}

int Simulator_gpu_dir::execution() {
		
//	if (options->verbose>0) { cout << endl << "--------------------------" << endl; cout << "Launching GPU code for phase 4" << endl; }
	pdp_out->print_dcba_phase(4);

	pdp_out->print_profiling_dcba_phase("Launching GPU code for phase 4");
	
	/* USING GPU KERNELS */
	uint cu_threads=CU_THREADS;
	uint cu_blocksx=options->num_environments;
	uint cu_blocksy=options->num_parallel_simulations;
	uint re_chunk=0;
	if (d_structures->env_rule_size < 32*options->num_environments) {
		re_chunk=d_structures->env_rule_size;
	}
	else {
		re_chunk=d_structures->env_rule_size/options->num_environments+
		(d_structures->env_rule_size%options->num_environments)==0?0:1;
	}
		
	
	/* Apply kernel for Phase 4 */
	if (runcomp) {
		//	if (options->verbose>0)	cout << endl << "Launching kernel for phase 4: ";
		pdp_out->print_profiling_dcba_microphase_name("Launching kernel for phase 4");
		sdkResetTimer(&counters.timer);
		sdkStartTimer(&counters.timer);
	}
	
	dim3 dimGrid (cu_blocksx, cu_blocksy);
	dim3 dimBlock (cu_threads);

	if(options->micro){
		//TODO: sort this and preaccumulate partitions
		if (pdp_out->will_print_dcba_phase())
			cout << "(using micro DCBA kernel)"<<endl;

		dim3 dimGrid (cu_blocksx, cu_blocksy);
		dim3 dimBlock (cu_threads);


		hipStreamSynchronize(execution_stream);
		getLastCudaError("pre kernel for phase 4 micro launch failure");

		int stream_to_go=0;
		int start_partition=0;
		//Accumulated size
		int partition_size=0;

		for(int i=0;i<options->num_partitions;i++){
			int part_size=accum_offset[i+1] - accum_offset[i];

			if(part_size>=cu_threads){
				if(start_partition!=i){
					//there was something already accumulated, launch it
					kernel_phase4_rules <<<dimGrid,dimBlock,0,streams[stream_to_go]>>> (d_structures->rule,
														d_structures->configuration, d_structures->rhs,
														d_structures->nr,
														structures->ruleblock.rule_idx[accum_offset[start_partition]],
														structures->ruleblock.rule_idx[accum_offset[i]]);

					stream_to_go++;
					if(stream_to_go==NUM_STREAMS)
						stream_to_go=0;

				}
				uint part_end=accum_offset[i+1];
				if(i+1==options->num_partitions){
					//If we have finished, append the rest (independent blocks)
					cout<<"last chunk"<<endl;
					part_end+= options->independent_ruleblocks;
				}
				//Large partition, launch independently
				kernel_phase4_rules <<<dimGrid,dimBlock,0,streams[stream_to_go]>>> (d_structures->rule,
										d_structures->configuration, d_structures->rhs,
										d_structures->nr,
										structures->ruleblock.rule_idx[accum_offset[i]],
										structures->ruleblock.rule_idx[part_end]);


				stream_to_go++;
				if(stream_to_go==NUM_STREAMS)
					stream_to_go=0;
				start_partition=i+1;
				partition_size=0;
			}else{
				//Small partition, accumulate parts
				if(part_size+partition_size >=cu_threads||i+1==options->num_partitions){
					//Enough accumulate (or last iteration), launch
					uint part_end=accum_offset[i+1];
					if(i+1==options->num_partitions){
						//If we have finished, append the rest (independent blocks)
						part_end+= options->independent_ruleblocks;
					}
					kernel_phase4_rules <<<dimGrid,dimBlock,0,streams[stream_to_go]>>> (d_structures->rule,
						d_structures->configuration, d_structures->rhs,
						d_structures->nr,
						structures->ruleblock.rule_idx[accum_offset[start_partition]],
						structures->ruleblock.rule_idx[part_end]
						                               );


					stream_to_go++;
					if(stream_to_go==NUM_STREAMS)
						stream_to_go=0;
					start_partition=i+1;
					partition_size=0;
				}else{
					//Accumulate
					partition_size+=part_size;

				}
			}

		}
		kernel_phase4_env <<<dimGrid,dimBlock,0,execution_stream>>> (d_structures->rule,
					d_structures->configuration, d_structures->rhs,
					d_structures->nr, re_chunk);
		for(int i=0;i<NUM_STREAMS;i++){
			hipStreamSynchronize(streams[i]);
		}
		hipStreamSynchronize(execution_stream);

		getLastCudaError("kernel for phase 2 micro launch failure");

	}else{

		kernel_phase4 <<<dimGrid,dimBlock,0,execution_stream>>> (d_structures->rule,
			d_structures->configuration, d_structures->rhs,
			d_structures->nr, d_structures->pi_rule_size,
			d_structures->pi_rule_size+d_structures->env_rule_size,
			re_chunk, *options);

	}

	if (runcomp) {
		hipStreamSynchronize(execution_stream);
		getLastCudaError("kernel for phase 4 launch failure");

		sdkStopTimer(&counters.timer);
		counters.timesp4gpu=sdkGetTimerValue(&counters.timer);
//		if (options->verbose>0)	cout << counters.timesp4gpu << "ms." << endl;
		pdp_out->print_profiling_dcba_microphase_result(counters.timesp4gpu);
	}


	/**************************************/
	/* PROFILING AND CHECK-OUT PROCEDURES */
	/**************************************/
	
//	if (options->verbose>1) {
//		/* Temporal testing procedure */
//		cout << "Checking data from GPU:" << endl;
//		d_cfg.multiset = new MULTIPLICITY[structures->configuration.multiset_size];
//
//		checkCudaErrors(hipMemcpy(d_cfg.multiset, d_structures->configuration.multiset, d_structures->configuration.multiset_size*sizeof(MULTIPLICITY), hipMemcpyDeviceToHost));
//
//		print_configuration(d_cfg);
//
//		delete []d_cfg.multiset;
//	}
		
	
	/*******************************/
	/* REPRODUCING CODE ON THE CPU */
	/*******************************/
	if (runcomp) 
		gold_execution();
		
//	if (runcomp && options->verbose>1) {
//		cout << "Checking data from CPU:" << endl;
//		print_configuration(structures->configuration);
//	}
	
	if (runcomp && pdp_out->will_print_dcba_phase())
		cout << endl << "Time for phase 4: GPU=" << counters.timesp4gpu << " ms, CPU=" << counters.timesp4cpu << " ms, speedup=" << counters.timesp4cpu/counters.timesp4gpu << "x" << endl;

	pdp_out->print_end_profiling_dcba_phase();

	return 0;
}


bool Simulator_gpu_dir::check_step_errors(){

	checkCudaErrors(hipMemcpyAsync(data_error, d_data_error, data_error_size*sizeof(uint), hipMemcpyDeviceToHost,execution_stream));

	hipStreamSynchronize(execution_stream);

	/* Checking mutual consistency */
	pdp_out->print_profiling_dcba_microphase_name("Checking mutual consistency");

	if (data_error[0]==CONSISTENCY_ERROR) {
		pdp_out->print_profiling_dcba_microphase_result(false);

		checkCudaErrors(hipMemcpy(this->abv, this->d_abv, this->abv_size*sizeof(ABV_T), hipMemcpyDeviceToHost));

		cout << "Found inconsistent blocks:" << endl;
		for (unsigned int sim=0; sim < options->num_parallel_simulations; sim++)
			for (unsigned int env=0; env < options->num_environments; env++)
				for (unsigned int membr=0; membr<options->num_membranes; membr++) {
					uint charge=data_error[1+sim*options->num_environments*options->num_membranes+env*options->num_membranes+membr];
					uint block=data_error[1+options->num_parallel_simulations*options->num_environments*options->num_membranes+
											sim*options->num_environments*options->num_membranes+env*options->num_membranes+membr];
					if (block!=UINT_MAX) {
						cout << "For sim " << sim << ", env " << env << ", membr " << membr <<
							" conflicts with charge " << charge << " for block " <<  block << endl;

						for (int blk=0; blk<options->num_rule_blocks; blk++) {
							uint am=GET_MEMBRANE(structures->ruleblock.membrane[blk]);
							char ch=GET_BETA(structures->ruleblock.membrane[blk]);
							if (is_active(blk,env,sim) && am==membr && ch==charge)
								cout << "   Possibly conflicted with " << blk << endl;
						}
					}
				}

		checkCudaErrors(hipMemset(d_data_error,0,data_error_size*sizeof(uint)));

		return true;
	}
	else
		pdp_out->print_profiling_dcba_microphase_result(true);


	/* Checking updating errors */
	pdp_out->print_profiling_dcba_microphase_name("Checking updating errors");

	if (data_error[0]==UPDATING_CONFIGURATION_ERROR) {
		pdp_out->print_profiling_dcba_microphase_result(false);

		cout << "Stopped. Found errors:" << endl;
		for (unsigned int sim=0; sim < options->num_parallel_simulations; sim++)
			for (unsigned int env=0; env < options->num_environments; env++)
				for (unsigned int membr=0; membr<options->num_membranes; membr++)
					cout << "For sim " << sim << ", env " << env << ", membr " << membr <<
					" error for block " << data_error[1+sim*options->num_environments*options->num_membranes+env*options->num_membranes+membr]-1 << endl;

		checkCudaErrors(hipMemset(d_data_error,0,data_error_size*sizeof(uint)));
		return true;
	}
	else pdp_out->print_profiling_dcba_microphase_result(true);

	return false;

}

/*******************************************/
/* Methods of the GPU wrapper for printing */
/*******************************************/

void PDP_Psystem_redix_out_std_gpuwrapper::retrieve_configuration() {
	checkCudaErrors(hipMemcpy(structures->configuration.multiset, d_structures->configuration.multiset, d_structures->configuration.multiset_size*sizeof(MULTIPLICITY), hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(structures->configuration.membrane, d_structures->configuration.membrane, d_structures->configuration.membrane_size*sizeof(CHARGE), hipMemcpyDeviceToHost));
}

void PDP_Psystem_redix_out_std_gpuwrapper::retrieve_block() {
	checkCudaErrors(hipMemcpy(structures->nb, d_structures->nb, d_structures->nb_size*sizeof(MULTIPLICITY), hipMemcpyDeviceToHost));
}

void PDP_Psystem_redix_out_std_gpuwrapper::retrieve_rule() {
	checkCudaErrors(hipMemcpy(structures->nr, d_structures->nr, d_structures->nr_size*sizeof(MULTIPLICITY), hipMemcpyDeviceToHost));
}

void PDP_Psystem_redix_out_std_gpuwrapper::print_profiling_table () {
    /* Output profiling information */
	/* Independently of the verbosity level */
	if (runcomp) {
		cout << endl << "-------------------------" << endl;
			cout << "Profiling mode enabled" << endl << "-------------------------" << endl;

		float totalselgpu=counters->timesp1gpu+counters->timesp2gpu+counters->timesp3gpu;
		float totalselcpu=counters->timesp1cpu+counters->timesp2cpu+counters->timesp3cpu;
		float totalexgpu=counters->timesp4gpu;
		float totalexcpu=counters->timesp4cpu;
		float totalgpu=totalselgpu+totalexgpu;
		float totalcpu=totalselcpu+totalexcpu;

		cout << "Time information summary:" << endl << "-------------------------" << endl;
		cout << "* Time in parts"
			<< endl << "Speedup on phases: phase1=" << counters->timesp1cpu/counters->timesp1gpu << "x, phase2=" << counters->timesp2cpu/counters->timesp2gpu << "x, phase3=" <<counters->timesp3cpu/counters->timesp3gpu << "x, phase4=" <<counters->timesp4cpu/counters->timesp4gpu << "x"
			<< endl << "Time of phases on GPU: phase1=" << counters->timesp1gpu << "ms, phase2=" << counters->timesp2gpu << "ms, phase3=" << counters->timesp3gpu << "ms, phase4=" << counters->timesp4gpu << "ms"
			<< endl << "Time of phases on CPU: phase1=" << counters->timesp1cpu << "ms, phase2=" << counters->timesp2cpu << "ms, phase3=" <<counters->timesp3cpu << "ms, phase4=" <<counters->timesp4cpu << "ms"
			<< endl << "Profiling GPU: phase1=" << counters->timesp1gpu*100.0/totalgpu << "%, phase2=" << counters->timesp2gpu*100.0/totalgpu << "%, phase3=" <<counters->timesp3gpu*100.0/totalgpu << "%, phase4=" <<counters->timesp4gpu*100.0/totalgpu << "%"
			<< endl << "Profiling CPU: phase1=" << counters->timesp1cpu*100.0/totalcpu << "%, phase2=" << counters->timesp2cpu*100.0/totalcpu << "%, phase3=" <<counters->timesp3cpu*100.0/totalcpu << "%, phase4=" <<counters->timesp4cpu*100.0/totalcpu << "%"
			<< endl << "Profiling selection GPU: phase1=" << counters->timesp1gpu*100.0/totalselgpu << "%, phase2=" << counters->timesp2gpu*100.0/totalselgpu << "%, phase3=" <<counters->timesp3gpu*100.0/totalselgpu << "%"
			<< endl << "Profiling selection CPU: phase1=" << counters->timesp1cpu*100.0/totalselcpu << "%, phase2=" << counters->timesp2cpu*100.0/totalselcpu << "%, phase3=" <<counters->timesp3cpu*100.0/totalselcpu << "%" << endl;

		cout << "* Total time" << endl
			<< "Total selection time: GPU=" << totalselgpu << "ms, " << "CPU=" << totalselcpu << "ms, "
			<< "Speedup=" << totalselcpu/totalselgpu << "x"
			<< endl << "Total execution time: GPU=" << totalexgpu << "ms, " << "CPU=" << totalexcpu << "ms"
			<< ", Speedup=" << totalexcpu/totalexgpu << "x"
			<< endl << "Total time: GPU=" << totalgpu << ", CPU=" << totalcpu << ", Speedup=" << totalcpu/totalgpu << "x"
			<< endl;
	}
}

void PDP_Psystem_redix_out_std_gpuwrapper::print_temporal_configuration () {
	if (!pdpout->will_print_temporal_configuration()) return;

	retrieve_configuration();

	for (int sim=0;sim<options->num_parallel_simulations;sim++) {
		pdpout->print_simulation(psb+sim);
		pdpout->print_temporal_configuration(sim);
	}
}

void PDP_Psystem_redix_out_std_gpuwrapper::print_profiling_dcba_phase (const char * message) {
	if (!runcomp && !pdpout->will_print_dcba_phase()) return;

	cout << endl << "--------------------------" << endl <<
			message << endl;
}

void PDP_Psystem_redix_out_std_gpuwrapper::print_end_profiling_dcba_phase () {
	if (!runcomp && !pdpout->will_print_dcba_phase()) return;

	cout << "--------------------------" << endl<<endl;
}

void PDP_Psystem_redix_out_std_gpuwrapper::print_profiling_dcba_microphase_name (const char * message) {
	if (!runcomp && !pdpout->will_print_dcba_phase()) return;

	cout << message << ": ";

	cout.flush();
}

// This function should be called after executing the microphase
void PDP_Psystem_redix_out_std_gpuwrapper::print_profiling_dcba_microphase_result (float time) {
	if (!runcomp && !pdpout->will_print_dcba_phase()) return;

	cout << time << "ms." << endl;

	cout.flush();
}

// This function should be called after executing the microphase
void PDP_Psystem_redix_out_std_gpuwrapper::print_profiling_dcba_microphase_result (bool result) {
	if (!runcomp && !pdpout->will_print_dcba_phase()) return;

	if (result) cout << "OK." << endl;
	else cout << "FAILED." << endl;

	cout.flush();
}

// This function should be called after executing the microphase
void PDP_Psystem_redix_out_std_gpuwrapper::print_profiling_dcba_microphase_datum (const char* message1, float datum, const char* message2) {
	if (!runcomp && !pdpout->will_print_dcba_phase()) return;

	cout << message1 << " " << datum << " " << message2 << endl;

	cout.flush();
}
// This function should be called after executing the microphase
void PDP_Psystem_redix_out_std_gpuwrapper::print_block_competition(int competing_block,bool env_blocks) {
	if (!runcomp && !pdpout->will_print_configuration()) return;
	pdpout->print_block_competition(competing_block,env_blocks);

}
